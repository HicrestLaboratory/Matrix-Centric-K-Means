#include "hip/hip_runtime.h"

#include "common.cuh"

#include <raft/core/device_mdarray.hpp>
#include <raft/cluster/kmeans.cuh>
#include <raft/cluster/kmeans_types.hpp>
#include <raft/core/resources.hpp>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include <ctime>




#include <cstdint>
#include <optional>

#define SEPARATOR ","
#define MAX_LINE   8192

int n_trials = 10;

namespace raft {
namespace cluster {
namespace detail {
template <typename DataT, typename IndexT>
void my_kmeans_fit_main(raft::resources const& handle,
                                         const cluster::KMeansParams& params,
                                         raft::device_matrix_view<const DataT, IndexT> X,
                                         raft::device_vector_view<const DataT, IndexT> weight,
                                         raft::device_matrix_view<DataT, IndexT> centroidsRawData,
                                         raft::host_scalar_view<DataT> inertia,
                                         raft::host_scalar_view<IndexT> n_iter,
                                         rmm::device_uvector<char>& workspace,
                                         bool check_converged=false)
{
    common::nvtx::range<common::nvtx::domain::raft> fun_scope("kmeans_fit_main");
    logger::get(RAFT_NAME).set_level(params.verbosity);
    hipStream_t stream = resource::get_cuda_stream(handle);
    auto n_samples          = X.extent(0);
    auto n_features         = X.extent(1);
    auto n_clusters         = params.n_clusters;
    auto metric                 = params.metric;

    // stores (key, value) pair corresponding to each sample where
    //   - key is the index of nearest cluster
    //   - value is the distance to the nearest cluster
    auto minClusterAndDistance =
        raft::make_device_vector<raft::KeyValuePair<IndexT, DataT>, IndexT>(handle, n_samples);

    // temporary buffer to store L2 norm of centroids or distance matrix,
    // destructor releases the resource
    rmm::device_uvector<DataT> L2NormBuf_OR_DistBuf(0, stream);

    // temporary buffer to store intermediate centroids, destructor releases the
    // resource
    auto newCentroids = raft::make_device_matrix<DataT, IndexT>(handle, n_clusters, n_features);

    // temporary buffer to store weights per cluster, destructor releases the
    // resource
    auto wtInCluster = raft::make_device_vector<DataT, IndexT>(handle, n_clusters);

    rmm::device_scalar<DataT> clusterCostD(stream);

    // L2 norm of X: ||x||^2
    auto L2NormX = raft::make_device_vector<DataT, IndexT>(handle, n_samples);
    auto l2normx_view =
        raft::make_device_vector_view<const DataT, IndexT>(L2NormX.data_handle(), n_samples);

    if (metric == raft::distance::DistanceType::L2Expanded ||
            metric == raft::distance::DistanceType::L2SqrtExpanded) {
        raft::linalg::rowNorm(L2NormX.data_handle(),
                                                    X.data_handle(),
                                                    X.extent(1),
                                                    X.extent(0),
                                                    raft::linalg::L2Norm,
                                                    true,
                                                    stream);
    }

    RAFT_LOG_DEBUG(
        "Calling KMeans.fit with %d samples of input data and the initialized "
        "cluster centers",
        n_samples);

    double update_time = 0.0;
    double dist_time = 0.0;
    IndexT d = X.extent(1);
    IndexT n = X.extent(0);
    IndexT k = n_clusters;

    DataT priorClusteringCost = 0;
#if LOG
    std::ofstream centroids_out;
    centroids_out.open("centroids-raft.out");
#endif
    for (n_iter[0] = 1; n_iter[0] <= params.max_iter; ++n_iter[0]) {
        RAFT_LOG_DEBUG(
            "KMeans.fit: Iteration-%d: fitting the model using the initialized "
            "cluster centers",
            n_iter[0]);
        //std::cout<<"iter "<<n_iter[0]<<std::endl;
#if LOG

        DataT * h_centroids = new DataT[d*k];
        raft::copy(h_centroids, centroidsRawData.data_handle(), k*d, stream);

        for (int i=0; i<k; i++) {
            for (int j=0; j<d; j++) {
                centroids_out<<h_centroids[d*i + j]<<",";
            }
            centroids_out<<std::endl;
        }


        delete[] h_centroids;
#endif

        auto centroids = raft::make_device_matrix_view<DataT, IndexT>(
            centroidsRawData.data_handle(), n_clusters, n_features);

        // computes minClusterAndDistance[0:n_samples) where
        // minClusterAndDistance[i] is a <key, value> pair where
        //   'key' is index to a sample in 'centroids' (index of the nearest
        //   centroid) and 'value' is the distance between the sample 'X[i]' and the
        //   'centroid[key]'
        auto stime_dist = std::chrono::system_clock::now();
        detail::minClusterAndDistanceCompute<DataT, IndexT>(handle,
                                                            X,
                                                            centroids,
                                                            minClusterAndDistance.view(),
                                                            l2normx_view,
                                                            L2NormBuf_OR_DistBuf,
                                                            params.metric,
                                                            params.batch_samples,
                                                            params.batch_centroids,
                                                            workspace);
        auto etime_dist = std::chrono::system_clock::now();
        auto dist_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime_dist-stime_dist);
        dist_time += dist_duration.count();

        // Using TransformInputIteratorT to dereference an array of
        // raft::KeyValuePair and converting them to just return the Key to be used
        // in reduce_rows_by_key prims
        detail::KeyValueIndexOp<IndexT, DataT> conversion_op;
        hipcub::TransformInputIterator<IndexT,
                                                                detail::KeyValueIndexOp<IndexT, DataT>,
                                                                raft::KeyValuePair<IndexT, DataT>*>
            itr(minClusterAndDistance.data_handle(), conversion_op);
#if LOG
        thrust::device_vector<uint32_t> d_clusters(n);
        thrust::copy(itr, itr+n, d_clusters.begin());
        uint32_t * h_clusters = new uint32_t[n];
        hipMemcpy(h_clusters, thrust::raw_pointer_cast(d_clusters.data()), sizeof(uint32_t)*n, hipMemcpyDeviceToHost);
        centroids_out<<"CLUSTERS"<<std::endl;
        for (int i=0; i<n; i++) {
            centroids_out<<h_clusters[i]<<",";
        }
        centroids_out<<std::endl;
        delete[] h_clusters;
#endif

        auto stime_update = std::chrono::system_clock::now();
        update_centroids(handle,
                                 X,
                                 weight,
                                 raft::make_device_matrix_view<const DataT, IndexT>(
                                     centroidsRawData.data_handle(), n_clusters, n_features),
                                 itr,
                                 wtInCluster.view(),
                                 newCentroids.view(),
                                 workspace);
        auto etime_update = std::chrono::system_clock::now();
        auto update_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime_update-stime_update);
        update_time += update_duration.count();

        // compute the squared norm between the newCentroids and the original
        // centroids, destructor releases the resource
        auto sqrdNorm = raft::make_device_scalar(handle, DataT(0));
        raft::linalg::mapThenSumReduce(sqrdNorm.data_handle(),
                                                                     newCentroids.size(),
                                                                     raft::sqdiff_op{},
                                                                     stream,
                                                                     centroids.data_handle(),
                                                                     newCentroids.data_handle());

        DataT sqrdNormError = 0;
        raft::copy(&sqrdNormError, sqrdNorm.data_handle(), sqrdNorm.size(), stream);

        raft::copy(
            centroidsRawData.data_handle(), newCentroids.data_handle(), newCentroids.size(), stream);


        bool done = false;
        if (params.inertia_check) {
            // calculate cluster cost phi_x(C)
            detail::computeClusterCost(handle,
                                                                 minClusterAndDistance.view(),
                                                                 workspace,
                                                                 raft::make_device_scalar_view(clusterCostD.data()),
                                                                 raft::value_op{},
                                                                 raft::add_op{});

            DataT curClusteringCost = clusterCostD.value(stream);

            ASSERT(curClusteringCost != (DataT)0.0,
                         "Too few points and centroids being found is getting 0 cost from "
                         "centers");

            if (n_iter[0] > 1) {
                DataT delta = curClusteringCost / priorClusteringCost;
                //if (delta > 1 - params.tol) done = true;
            }
            priorClusteringCost = curClusteringCost;
        }

        resource::sync_stream(handle, stream);
        if ((check_converged) && sqrdNormError < params.tol) done = true;

        if (done) {
            RAFT_LOG_DEBUG("Threshold triggered after %d iterations. Terminating early.", n_iter[0]);
            break;
        }
#if LOG
        centroids_out<<"END ITERATION "<<(n_iter[0]-1)<<std::endl;
#endif
    }
#if LOG
    centroids_out.close();
#endif

    n_iter[0]--;

    std::cout<<std::fixed<<"centroids-update-time: "<<update_time<<"s"<<std::endl;
    std::cout<<std::fixed<<"dist-argmin-time: "<<dist_time<<"s"<<std::endl;
    std::cout<<"n_iter[0] "<<n_iter[0]<<std::endl;

    auto centroids = raft::make_device_matrix_view<DataT, IndexT>(
        centroidsRawData.data_handle(), n_clusters, n_features);

    detail::minClusterAndDistanceCompute<DataT, IndexT>(handle,
                                                                                                            X,
                                                                                                            centroids,
                                                                                                            minClusterAndDistance.view(),
                                                                                                            l2normx_view,
                                                                                                            L2NormBuf_OR_DistBuf,
                                                                                                            params.metric,
                                                                                                            params.batch_samples,
                                                                                                            params.batch_centroids,
                                                                                                            workspace);

    // TODO: add different templates for InType of binaryOp to avoid thrust transform
    thrust::transform(resource::get_thrust_policy(handle),
                                        minClusterAndDistance.data_handle(),
                                        minClusterAndDistance.data_handle() + minClusterAndDistance.size(),
                                        weight.data_handle(),
                                        minClusterAndDistance.data_handle(),
                                        [=] __device__(const raft::KeyValuePair<IndexT, DataT> kvp, DataT wt) {
                                            raft::KeyValuePair<IndexT, DataT> res;
                                            res.value = kvp.value * wt;
                                            res.key     = kvp.key;
                                            return res;
                                        });

    // calculate cluster cost phi_x(C)
    detail::computeClusterCost(handle,
                                                         minClusterAndDistance.view(),
                                                         workspace,
                                                         raft::make_device_scalar_view(clusterCostD.data()),
                                                         raft::value_op{},
                                                         raft::add_op{});

    inertia[0] = clusterCostD.value(stream);

    RAFT_LOG_DEBUG("KMeans.fit: completed after %d iterations with %f inertia[0] ",
                                 n_iter[0] > params.max_iter ? n_iter[0] - 1 : n_iter[0],
                                 inertia[0]);
}
}
}
}



template <typename T>
void read_svm(const uint32_t n, const uint32_t d,
                std::ifstream& in, T * d_dataset)
{
    T * h_dataset = new T[d*n];

    std::string str;

    int i = 0;
    while (std::getline(in, str, '\n')) {
        std::istringstream input_str(str);
        std::string token;
        while (std::getline(input_str, token, ' ')) {
            std::istringstream token_stream(token);
            std::string key, value;
            if (std::getline(token_stream, key, ':') &&
                std::getline(token_stream, value)) {
                    h_dataset[std::atoi(key.c_str())-1 + d*i] = std::atof(value.c_str());
            }
        }
        i++;
    }


    hipMemcpy(d_dataset, h_dataset, sizeof(T)*d*n, hipMemcpyHostToDevice);
	delete[] h_dataset;

}


template <typename T>
void read_csv(const uint32_t n, const uint32_t d,
				std::istream& in, T * d_dataset)
{

    T * h_dataset = new T[d*n];

    std::string str;

    int i = 0;
    while (std::getline(in, str, '\n')) {

        /* Skip header */
        if (i==0) {
            i++;
            continue;
        }

        std::istringstream input_str(str);
        std::string token;
        int j = 0;
        while (std::getline(input_str, token, ',')) {

            /* Skip class label */
            if (j==0) {
                j++;
                continue;
            }

            std::istringstream token_stream(token);
            h_dataset[j-1 + d*(i-1)] = std::atof(token.c_str());
            j++;
        }
        i++;
    }
    /*
    for (size_t i = 0; i <= n; i++) {
    char str[MAX_LINE] = { 0 };
    in >> str;

    if (i == (size_t)0) { continue; }
    if (!str[0]) { break; }

    int j = 0;
    char *tok = strtok(str, SEPARATOR);
    while (tok && j < d) {
      h_dataset[j + (i-1)*d] = atof(tok);
      tok = strtok(NULL, SEPARATOR);
      j++;
    }

    }
  */

    /*
    std::ofstream points_out;
    points_out.open("points-raft.out");
    for (int i=0; i<n; i++) {
      for (int j=0; j<d; j++) {
          points_out<<h_dataset[j + i*d]<<",";
      }
      points_out<<std::endl;
    }
    points_out.close();
    */

    hipMemcpy(d_dataset, h_dataset, sizeof(T)*d*n, hipMemcpyHostToDevice);
    delete[] h_dataset;
}



void run_kmeans(const uint32_t n, const uint32_t d, const uint32_t k, const uint32_t n_iters, const bool check_converged,
                float tol,
                std::string infile)
{
    using namespace raft;
    
    typedef float data_t; 
    typedef uint32_t ind_t;

    std::ifstream istream;
    istream.open(infile);

	std::random_device rd;
	std::mt19937 gen(rd());


    const raft::resources handle;
    cluster::KMeansParams params;
    params.n_clusters = k;
    params.max_iter = n_iters;
    params.init = cluster::KMeansParams::InitMethod::Random;
    params.tol = tol;
    params.inertia_check = true;

    auto centroids = raft::make_device_matrix<data_t, ind_t>(handle, k, d);
    auto points = raft::make_device_matrix<data_t, ind_t>(handle, n, d);


    if (infile.compare("-1")==0) { 
        raft::random::RngState rand(1234ULL);
        raft::random::uniform(handle, rand,
                        raft::make_device_vector_view(points.data_handle(), points.size()),
                        (data_t)-1e5, (data_t)1e5);
    } else if (infile.find("svm")!=std::string::npos) {
        read_svm(n, d, istream, points.data_handle());
    } else if (infile.find("csv")!=std::string::npos) {
        read_csv(n, d, istream, points.data_handle());
    }

    istream.close();


    auto points_view = raft::make_device_matrix_view<const data_t>(points.data_handle(),
                                                                    n, d);
    auto weight = raft::make_device_vector<data_t, ind_t>(handle, n);
    thrust::fill(raft::resource::get_thrust_policy(handle),
                    weight.data_handle(),
                    weight.data_handle() + weight.size(),
                    1);

    hipStream_t stream = raft::resource::get_cuda_stream(handle);
    rmm::device_uvector<char> workspace(0, stream);

    data_t inertia; ind_t n_iter_run;

    std::cout<<"Running kmeans"<<std::endl;
    std::cout<<"n:"<<n<<" d:"<<d<<" k:"<<k<<std::endl;

    double score = 0;

    double kmeans_time = 0;
    for (int i=0; i<n_trials; i++) {
        params.rng_state.seed = gen();
        cluster::detail::initRandom<data_t, ind_t>(handle, params, points_view, centroids.view());

        auto stime = std::chrono::system_clock::now();
        cluster::detail::my_kmeans_fit_main<data_t, ind_t>
                            (handle,
                             params,
                             points_view,
                             weight.view(),
                             centroids.view(),
                             raft::make_host_scalar_view(&inertia),
                             raft::make_host_scalar_view(&n_iter_run),
                             workspace,
                             check_converged);
        auto etime = std::chrono::system_clock::now();
        auto kmeans_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime - stime);
        if (i>0) {
            kmeans_time += kmeans_duration.count();
        }
        score += inertia;
        std::cout<<inertia<<std::endl;
    }

    kmeans_time /= (n_trials - 1);
    score /= (n_trials);



    double fused_dist_time = 0;
    /*
    {
        auto minClusterAndDistance = raft::make_device_vector<raft::KeyValuePair<ind_t, data_t>, ind_t>(handle, n);

        rmm::device_uvector<data_t> buf(0, stream);

        // Run fused distances-argmin kernel
        for (int i=0; i<n_iter_run; i++) {
            auto stime = std::chrono::system_clock::now();

            auto l2Norm = raft::make_device_vector<data_t, ind_t>(handle, n);
            linalg::rowNorm(l2Norm.data_handle(), points.data_handle(), points.extent(1), points.extent(0),
                            linalg::L2Norm, true, stream); 
            auto l2Norm_view = raft::make_device_vector_view<const data_t>(l2Norm.data_handle(),
                                                                        n);
            cluster::detail::minClusterAndDistanceCompute<data_t, ind_t>
                    (
                    handle,
                    points_view,
                    centroids.view(),
                    minClusterAndDistance.view(),
                    l2Norm_view,
                    buf,
                    distance::DistanceType::L2Expanded,
                    n,
                    k,
                    workspace);
            resource::sync_stream(handle, stream);
            auto etime = std::chrono::system_clock::now();
            auto fused_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime - stime);
            fused_dist_time += fused_duration.count();
        }

    }*/

    // Run distances and argmin separately
    double pw_dist_time = 0;
    double argmin_time = 0;
    /*
    {

        auto pwDist = raft::make_device_matrix<data_t, ind_t>(handle, n, k);

        auto minClusterAndDistance = raft::make_device_vector<raft::KeyValuePair<ind_t, data_t>, ind_t>(handle, n);

        raft::KeyValuePair<ind_t, data_t> init(0, std::numeric_limits<data_t>::max());

        for (int i=0; i<n_iter_run; i++) {
            std::cout<<"Iteration "<<i<<std::endl;
            auto stime = std::chrono::system_clock::now();
            cluster::detail::pairwise_distance_kmeans<data_t, ind_t>
                    (
                    handle,
                    points_view,
                    centroids.view(),
                    pwDist.view(),
                    workspace,
                    distance::DistanceType::L2Expanded);
            resource::sync_stream(handle, stream);
            auto etime = std::chrono::system_clock::now();
            auto pw_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime - stime);
            pw_dist_time += pw_duration.count();

            stime = std::chrono::system_clock::now();
            linalg::coalescedReduction
                                        (minClusterAndDistance.data_handle(),
                                        pwDist.data_handle(),
                                        pwDist.extent(1), pwDist.extent(0),
                                        init,
                                        stream, true,
                                        [=] __device__(const data_t val, const ind_t i) {
                                            raft::KeyValuePair<ind_t, data_t> pair;
                                            pair.key   = i;
                                            pair.value = val;
                                            return pair;
                                        },
                                        raft::argmin_op{},
                                        raft::identity_op{});
            etime = std::chrono::system_clock::now();
            auto amin_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime - stime);
            argmin_time += amin_duration.count();
        }
    }

    */

    std::cout<<std::fixed<<"kmeans-time: "<<kmeans_time<<"s"<<std::endl;
    std::cout<<std::fixed<<"kmeans-score: "<<score<<std::endl;
    std::cout<<std::fixed<<"kmeans-iterations: "<<n_iter_run<<std::endl;
                                        
}



int main(int argc, char ** argv)
{
    int n = std::atoi(argv[1]);
    int d = std::atoi(argv[2]);
    int k = std::atoi(argv[3]);
    int n_iters = std::atoi(argv[4]);
    bool check_converged = (bool)std::atoi(argv[5]);
    std::string infile;
    if (argc > 6)
        infile = std::string(argv[6]);
    else
        infile = std::string("-1");
    run_kmeans(n, d, k, n_iters, check_converged,0.0001, infile);
    return 0;
}
