#include "hip/hip_runtime.h"

#include "common.cuh"

#include <raft/core/device_mdarray.hpp>
#include <raft/cluster/kmeans.cuh>
#include <raft/cluster/kmeans_types.hpp>
#include <raft/core/resources.hpp>




#include <cstdint>
#include <optional>

#define N_ITERS 2 

namespace raft {
namespace cluster {
namespace detail {
template <typename DataT, typename IndexT>
void my_kmeans_fit_main(raft::resources const& handle,
                                         const cluster::KMeansParams& params,
                                         raft::device_matrix_view<const DataT, IndexT> X,
                                         raft::device_vector_view<const DataT, IndexT> weight,
                                         raft::device_matrix_view<DataT, IndexT> centroidsRawData,
                                         raft::host_scalar_view<DataT> inertia,
                                         raft::host_scalar_view<IndexT> n_iter,
                                         rmm::device_uvector<char>& workspace)
{
    common::nvtx::range<common::nvtx::domain::raft> fun_scope("kmeans_fit_main");
    logger::get(RAFT_NAME).set_level(params.verbosity);
    hipStream_t stream = resource::get_cuda_stream(handle);
    auto n_samples          = X.extent(0);
    auto n_features         = X.extent(1);
    auto n_clusters         = params.n_clusters;
    auto metric                 = params.metric;

    // stores (key, value) pair corresponding to each sample where
    //   - key is the index of nearest cluster
    //   - value is the distance to the nearest cluster
    auto minClusterAndDistance =
        raft::make_device_vector<raft::KeyValuePair<IndexT, DataT>, IndexT>(handle, n_samples);

    // temporary buffer to store L2 norm of centroids or distance matrix,
    // destructor releases the resource
    rmm::device_uvector<DataT> L2NormBuf_OR_DistBuf(0, stream);

    // temporary buffer to store intermediate centroids, destructor releases the
    // resource
    auto newCentroids = raft::make_device_matrix<DataT, IndexT>(handle, n_clusters, n_features);

    // temporary buffer to store weights per cluster, destructor releases the
    // resource
    auto wtInCluster = raft::make_device_vector<DataT, IndexT>(handle, n_clusters);

    rmm::device_scalar<DataT> clusterCostD(stream);

    // L2 norm of X: ||x||^2
    auto L2NormX = raft::make_device_vector<DataT, IndexT>(handle, n_samples);
    auto l2normx_view =
        raft::make_device_vector_view<const DataT, IndexT>(L2NormX.data_handle(), n_samples);

    if (metric == raft::distance::DistanceType::L2Expanded ||
            metric == raft::distance::DistanceType::L2SqrtExpanded) {
        raft::linalg::rowNorm(L2NormX.data_handle(),
                                                    X.data_handle(),
                                                    X.extent(1),
                                                    X.extent(0),
                                                    raft::linalg::L2Norm,
                                                    true,
                                                    stream);
    }

    RAFT_LOG_DEBUG(
        "Calling KMeans.fit with %d samples of input data and the initialized "
        "cluster centers",
        n_samples);

    double update_time = 0.0;
    double dist_time = 0.0;

    DataT priorClusteringCost = 0;
    for (n_iter[0] = 1; n_iter[0] <= params.max_iter; ++n_iter[0]) {
        RAFT_LOG_DEBUG(
            "KMeans.fit: Iteration-%d: fitting the model using the initialized "
            "cluster centers",
            n_iter[0]);
        std::cout<<"iter "<<n_iter[0]<<std::endl;

        auto centroids = raft::make_device_matrix_view<DataT, IndexT>(
            centroidsRawData.data_handle(), n_clusters, n_features);

        // computes minClusterAndDistance[0:n_samples) where
        // minClusterAndDistance[i] is a <key, value> pair where
        //   'key' is index to a sample in 'centroids' (index of the nearest
        //   centroid) and 'value' is the distance between the sample 'X[i]' and the
        //   'centroid[key]'
        auto stime_dist = std::chrono::system_clock::now();
        detail::minClusterAndDistanceCompute<DataT, IndexT>(handle,
                                                            X,
                                                            centroids,
                                                            minClusterAndDistance.view(),
                                                            l2normx_view,
                                                            L2NormBuf_OR_DistBuf,
                                                            params.metric,
                                                            params.batch_samples,
                                                            params.batch_centroids,
                                                            workspace);
        resource::sync_stream(handle, stream);
        auto etime_dist = std::chrono::system_clock::now();
        auto dist_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime_dist-stime_dist);
        dist_time += dist_duration.count();

        // Using TransformInputIteratorT to dereference an array of
        // raft::KeyValuePair and converting them to just return the Key to be used
        // in reduce_rows_by_key prims
        detail::KeyValueIndexOp<IndexT, DataT> conversion_op;
        hipcub::TransformInputIterator<IndexT,
                                                                detail::KeyValueIndexOp<IndexT, DataT>,
                                                                raft::KeyValuePair<IndexT, DataT>*>
            itr(minClusterAndDistance.data_handle(), conversion_op);

        auto stime_update = std::chrono::system_clock::now();
        update_centroids(handle,
                                 X,
                                 weight,
                                 raft::make_device_matrix_view<const DataT, IndexT>(
                                     centroidsRawData.data_handle(), n_clusters, n_features),
                                 itr,
                                 wtInCluster.view(),
                                 newCentroids.view(),
                                 workspace);
        resource::sync_stream(handle, stream);
        auto etime_update = std::chrono::system_clock::now();
        auto update_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime_update-stime_update);
        update_time += update_duration.count();

        // compute the squared norm between the newCentroids and the original
        // centroids, destructor releases the resource
        auto sqrdNorm = raft::make_device_scalar(handle, DataT(0));
        raft::linalg::mapThenSumReduce(sqrdNorm.data_handle(),
                                                                     newCentroids.size(),
                                                                     raft::sqdiff_op{},
                                                                     stream,
                                                                     centroids.data_handle(),
                                                                     newCentroids.data_handle());

        DataT sqrdNormError = 0;
        raft::copy(&sqrdNormError, sqrdNorm.data_handle(), sqrdNorm.size(), stream);

        raft::copy(
            centroidsRawData.data_handle(), newCentroids.data_handle(), newCentroids.size(), stream);

        bool done = false;
        if (params.inertia_check) {
            // calculate cluster cost phi_x(C)
            detail::computeClusterCost(handle,
                                                                 minClusterAndDistance.view(),
                                                                 workspace,
                                                                 raft::make_device_scalar_view(clusterCostD.data()),
                                                                 raft::value_op{},
                                                                 raft::add_op{});

            DataT curClusteringCost = clusterCostD.value(stream);

            ASSERT(curClusteringCost != (DataT)0.0,
                         "Too few points and centroids being found is getting 0 cost from "
                         "centers");

            if (n_iter[0] > 1) {
                DataT delta = curClusteringCost / priorClusteringCost;
                //if (delta > 1 - params.tol) done = true;
                //make sure we run for all iters
            }
            priorClusteringCost = curClusteringCost;
        }

        resource::sync_stream(handle, stream);
        if (sqrdNormError < params.tol) done = true;

        if (done) {
            RAFT_LOG_DEBUG("Threshold triggered after %d iterations. Terminating early.", n_iter[0]);
            break;
        }
    }

    n_iter[0]--;

    std::cout<<"centroids-update-time: "<<update_time/n_iter[0]<<"s"<<std::endl;
    std::cout<<"n_iter[0] "<<n_iter[0]<<std::endl;

    auto centroids = raft::make_device_matrix_view<DataT, IndexT>(
        centroidsRawData.data_handle(), n_clusters, n_features);

    detail::minClusterAndDistanceCompute<DataT, IndexT>(handle,
                                                                                                            X,
                                                                                                            centroids,
                                                                                                            minClusterAndDistance.view(),
                                                                                                            l2normx_view,
                                                                                                            L2NormBuf_OR_DistBuf,
                                                                                                            params.metric,
                                                                                                            params.batch_samples,
                                                                                                            params.batch_centroids,
                                                                                                            workspace);

    // TODO: add different templates for InType of binaryOp to avoid thrust transform
    thrust::transform(resource::get_thrust_policy(handle),
                                        minClusterAndDistance.data_handle(),
                                        minClusterAndDistance.data_handle() + minClusterAndDistance.size(),
                                        weight.data_handle(),
                                        minClusterAndDistance.data_handle(),
                                        [=] __device__(const raft::KeyValuePair<IndexT, DataT> kvp, DataT wt) {
                                            raft::KeyValuePair<IndexT, DataT> res;
                                            res.value = kvp.value * wt;
                                            res.key     = kvp.key;
                                            return res;
                                        });

    // calculate cluster cost phi_x(C)
    detail::computeClusterCost(handle,
                                                         minClusterAndDistance.view(),
                                                         workspace,
                                                         raft::make_device_scalar_view(clusterCostD.data()),
                                                         raft::value_op{},
                                                         raft::add_op{});

    inertia[0] = clusterCostD.value(stream);

    RAFT_LOG_DEBUG("KMeans.fit: completed after %d iterations with %f inertia[0] ",
                                 n_iter[0] > params.max_iter ? n_iter[0] - 1 : n_iter[0],
                                 inertia[0]);
}
}
}
}








void run_kmeans(const uint32_t n, const uint32_t d, const uint32_t k)
{
    
    typedef float data_t; 
    typedef uint32_t ind_t;

    using namespace raft;

    const raft::resources handle;
    cluster::KMeansParams params;
    params.n_clusters = k;
    params.max_iter = N_ITERS;
    params.init = cluster::KMeansParams::InitMethod::Random;

    auto centroids = raft::make_device_matrix<data_t, ind_t>(handle, k, d);
    auto points = raft::make_device_matrix<data_t, ind_t>(handle, n, d);

    raft::random::RngState rand(1234ULL);
    raft::random::uniform(handle, rand,
                    raft::make_device_vector_view(points.data_handle(), points.size()),
                    -1.0f, 1.0f);

    auto points_view = raft::make_device_matrix_view<const data_t>(points.data_handle(),
                                                                    n, d);
    auto weight = raft::make_device_vector<data_t, ind_t>(handle, n);
    thrust::fill(raft::resource::get_thrust_policy(handle),
                    weight.data_handle(),
                    weight.data_handle() + weight.size(),
                    1);

    hipStream_t stream = raft::resource::get_cuda_stream(handle);
    rmm::device_uvector<char> workspace(0, stream);

    data_t inertia; ind_t n_iter_run;

    std::cout<<"Running kmeans"<<std::endl;
    std::cout<<"n:"<<n<<" d:"<<d<<" k:"<<k<<std::endl;

    auto stime = std::chrono::system_clock::now();
    cluster::detail::my_kmeans_fit_main<data_t, ind_t>
                        (handle,
                         params,
                         points_view,
                         weight.view(),
                         centroids.view(),
                         raft::make_host_scalar_view(&inertia),
                         raft::make_host_scalar_view(&n_iter_run),
                         workspace);
    auto etime = std::chrono::system_clock::now();

    double fused_dist_time = 0;

    auto kmeans_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime - stime);
    {
        auto minClusterAndDistance = raft::make_device_vector<raft::KeyValuePair<ind_t, data_t>, ind_t>(handle, n);

        auto l2Norm = raft::make_device_vector<data_t, ind_t>(handle, n);
        linalg::rowNorm(l2Norm.data_handle(), points.data_handle(), points.extent(1), points.extent(0),
                        linalg::L2Norm, true, stream); 
        auto l2Norm_view = raft::make_device_vector_view<const data_t>(l2Norm.data_handle(),
                                                                        n);
        rmm::device_uvector<data_t> buf(0, stream);

        // Run fused distances-argmin kernel
        for (int i=0; i<n_iter_run; i++) {
            stime = std::chrono::system_clock::now();
            cluster::detail::minClusterAndDistanceCompute<data_t, ind_t>
                    (
                    handle,
                    points_view,
                    centroids.view(),
                    minClusterAndDistance.view(),
                    l2Norm_view,
                    buf,
                    distance::DistanceType::L2Expanded,
                    n,
                    k,
                    workspace);
            resource::sync_stream(handle, stream);
            etime = std::chrono::system_clock::now();
            auto fused_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime - stime);
            fused_dist_time += fused_duration.count();
        }

    }

    // Run distances and argmin separately
    double pw_dist_time = 0;
    double argmin_time = 0;
    {

        auto pwDist = raft::make_device_matrix<data_t, ind_t>(handle, n, k);

        auto minClusterAndDistance = raft::make_device_vector<raft::KeyValuePair<ind_t, data_t>, ind_t>(handle, n);

        raft::KeyValuePair<ind_t, data_t> init(0, std::numeric_limits<data_t>::max());

        for (int i=0; i<n_iter_run; i++) {
            stime = std::chrono::system_clock::now();
            cluster::detail::pairwise_distance_kmeans<data_t, ind_t>
                    (
                    handle,
                    points_view,
                    centroids.view(),
                    pwDist.view(),
                    workspace,
                    distance::DistanceType::L2Expanded);
            resource::sync_stream(handle, stream);
            etime = std::chrono::system_clock::now();
            auto pw_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime - stime);
            pw_dist_time += pw_duration.count();

            stime = std::chrono::system_clock::now();
            linalg::coalescedReduction
                                        (minClusterAndDistance.data_handle(),
                                        pwDist.data_handle(),
                                        pwDist.extent(1), pwDist.extent(0),
                                        init,
                                        stream, true,
                                        [=] __device__(const data_t val, const ind_t i) {
                                            raft::KeyValuePair<ind_t, data_t> pair;
                                            pair.key   = i;
                                            pair.value = val;
                                            return pair;
                                        },
                                        raft::argmin_op{},
                                        raft::identity_op{});
            etime = std::chrono::system_clock::now();
            auto amin_duration = std::chrono::duration_cast<std::chrono::duration<double>>(etime - stime);
            argmin_time += amin_duration.count();
        }
    }


    std::cout<<"kmeans-time: "<<kmeans_duration.count()<<"s"<<std::endl;
    std::cout<<"fused-dist-argmin-time: "<<fused_dist_time/n_iter_run<<"s"<<std::endl;
    std::cout<<"pwdist-time: "<<pw_dist_time/n_iter_run<<"s"<<std::endl;
    std::cout<<"argmin-time: "<<argmin_time/n_iter_run<<"s"<<std::endl;
    std::cout<<"kmeans-score: "<<inertia<<std::endl;
    std::cout<<"kmeans-iterations: "<<n_iter_run<<std::endl;
                                        
}



int main(int argc, char ** argv)
{
    int n = std::atoi(argv[1]);
    int d = std::atoi(argv[2]);
    int k = std::atoi(argv[3]);
    run_kmeans(n, d, k);
    return 0;
}
