#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <random>
#include <iomanip>
#include <hipcub/hipcub.hpp>
#include <cmath>
#include <limits>

#include "include/common.h"
#include "include/colors.h"

#include "utils.cuh"
#include "kmeans.cuh"

#include "kernels/argmin.cuh"

#define SHFL_MASK 0xFFFFFFFF

using namespace std;

random_device rd;
seed_seq seed{0}; // FIXME use rd()
mt19937 rng(seed);

// This function is not decalred in utils.cu because of compilation problems with __host__ __device__
__host__ __device__ unsigned int next_pow_2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

/*  DISTANCES KERNELS  */
__global__ void compute_distances_one_point_per_warp(DATA_TYPE* distances, DATA_TYPE* centroids, DATA_TYPE* points) {
  const uint64_t point_offset = blockIdx.x * blockDim.x + threadIdx.x;
  const uint64_t center_offset = blockIdx.y * blockDim.x + threadIdx.x;
  DATA_TYPE dist = points[point_offset] - centroids[center_offset];
  dist *= dist;
  
  for (int i = next_pow_2(blockDim.x); i > 0; i /= 2)
    dist += __shfl_down_sync(SHFL_MASK, dist, i);

  if (threadIdx.x == 0) {
    distances[(blockIdx.x * gridDim.y) + blockIdx.y] = dist;
  }
}

__global__ void compute_distances_shmem(DATA_TYPE* distances, DATA_TYPE* centroids, DATA_TYPE* points, const uint32_t points_per_warp, const uint32_t d) {
  const uint64_t point_i = (blockIdx.x * points_per_warp) + (threadIdx.x / d);
  const uint64_t center_i = blockIdx.y;
  const uint32_t d_i = threadIdx.x % d;
  const uint64_t dists_i = (center_i * blockDim.y * d) + ((point_i % points_per_warp) * d) + d_i;

  extern __shared__ DATA_TYPE dists[];

  if (threadIdx.x < points_per_warp * d) {
    DATA_TYPE dist = fabs(points[point_i * d + d_i] - centroids[center_i * d + d_i]);
    dists[dists_i] = dist * dist;
    __syncthreads();
    if (d_i == 0) {
      for (int i = 1; i < d; i++) {
        dists[dists_i] += dists[dists_i + i];
      }
      distances[(point_i * center_i) + point_i] = dists[dists_i];
    }
  }
}

__global__ void compute_distances_shfl(DATA_TYPE* distances, DATA_TYPE* centroids, DATA_TYPE* points, const uint32_t points_n, const uint32_t points_per_warp, const uint32_t d, const uint32_t d_closest_2_pow) {
  const uint64_t point_i = (blockIdx.x * points_per_warp) + (threadIdx.x / d_closest_2_pow);
  const uint64_t center_i = blockIdx.y;
  const uint32_t d_i = threadIdx.x % d_closest_2_pow;

  if (point_i < points_n && d_i < d) {
    DATA_TYPE dist = fabs(points[point_i * d + d_i] - centroids[center_i * d + d_i]);
    dist *= dist;
    for (int i = d_closest_2_pow / 2; i > 0; i /= 2) {
      dist += __shfl_down_sync(SHFL_MASK, dist, i);
      // if (point_i == 3) printf("%d  p: %lu c: %lu d: %u v: %.3f\n", i, point_i, center_i, d_i, dist);
    }
    if (d_i == 0) {
      distances[(point_i * gridDim.y) + center_i] = dist;
    }
  }
}

/*  CENTERS KERNELS  */
__global__ void compute_centroids_shfl(DATA_TYPE* centroids, DATA_TYPE* points, uint32_t* points_clusters, uint32_t* clusters_len, uint64_t n, uint32_t d) {  
  uint32_t cluster_idx = 2 * blockIdx.y * blockDim.x + threadIdx.x;
  uint32_t point_idx   = cluster_idx * blockDim.y + threadIdx.y;
  uint32_t cluster_off = blockDim.x;
  uint32_t point_off   = cluster_off * blockDim.y;
  
  float val = 0;

  if (point_idx < n * d && blockIdx.x == points_clusters[cluster_idx]) { 
    val = points[point_idx]; 
  }
  
  if (point_idx + point_off < n * d && blockIdx.x == points_clusters[cluster_idx + cluster_off]) { 
    val += points[point_idx + point_off]; 
  } 
  
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    val += __shfl_down_sync(SHFL_MASK, val, offset);
  }
  
  if (threadIdx.x % warpSize == 0) {
    uint32_t count = clusters_len[blockIdx.x] > 1 ? clusters_len[blockIdx.x] : 1; 
    DATA_TYPE scale = 1.0 / ((double) count); 
    val *= scale;   
    atomicAdd(&centroids[blockIdx.x * blockDim.y + threadIdx.y], val);
  }
}

__global__ void compute_centroids_shfl_shrd(DATA_TYPE* centroids, DATA_TYPE* points, uint32_t* points_clusters, uint32_t* clusters_len, uint64_t n, uint32_t d) {  
  uint32_t cluster_idx = 2 * blockIdx.y * blockDim.x + threadIdx.x;
  uint32_t point_idx   = cluster_idx * blockDim.y + threadIdx.y;
  uint32_t cluster_off = blockDim.x;
  uint32_t point_off   = cluster_off * blockDim.y;
  
  float val = 0;
  extern __shared__ DATA_TYPE shrd_mem[];

  if (point_idx < n * d && blockIdx.x == points_clusters[cluster_idx]) { 
    val = points[point_idx]; 
  }
  
  if (point_idx + point_off < n * d && blockIdx.x == points_clusters[cluster_idx + cluster_off]) { 
    val += points[point_idx + point_off]; 
  }
  
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    val += __shfl_down_sync(SHFL_MASK, val, offset);
  }
  
  if (threadIdx.x % warpSize == 0) {
    uint32_t warp_idx   = threadIdx.x / warpSize;
    uint32_t shrd_dim_y = blockDim.x  / warpSize;
    uint32_t shrd_idx   = threadIdx.y * shrd_dim_y + warp_idx;
    
    shrd_mem[shrd_idx] = val;
    __syncthreads();
    
    for (int offset = shrd_dim_y / 2; offset > 0; offset /= 2) {
      if (warp_idx < offset) {
        shrd_mem[shrd_idx] += shrd_mem[shrd_idx + offset];
      }
      __syncthreads();
    }

    if (shrd_idx % shrd_dim_y == 0) {
      uint32_t count = clusters_len[blockIdx.x] > 1 ? clusters_len[blockIdx.x] : 1; 
      DATA_TYPE scale = 1.0 / ((double) count); 
      val = shrd_mem[shrd_idx] * scale;   
      atomicAdd(&centroids[blockIdx.x * blockDim.y + threadIdx.y], val);
    }
  }
}

/* Kmeans class */
void Kmeans::init_centroids (Point<DATA_TYPE>** points) {
  uniform_int_distribution<int> random_int(0, n - 1);
  CHECK_CUDA_ERROR(hipHostAlloc(&h_centroids, CENTROIDS_BYTES, hipHostMallocDefault));
  CHECK_CUDA_ERROR(hipHostAlloc(&h_last_centroids, CENTROIDS_BYTES, hipHostMallocDefault));
  unsigned int i = 0;
  vector<Point<DATA_TYPE>*> usedPoints;
  Point<DATA_TYPE>* centroids[k];
  while (i < k) {
    Point<DATA_TYPE>* p = points[random_int(rng)];
    bool found = false;
    for (auto p1 : usedPoints) {
      if ((*p1) == (*p)) { // FIXME Is it better use some min distance??
        found = true;
        break;
      }
    }
    if (!found) {
      for (unsigned int j = 0; j < d; ++j) {
        h_centroids[i * d + j] = p->get(j);
      }
      centroids[i] = new Point<DATA_TYPE>(p);
      usedPoints.push_back(p);
      ++i;
    }
  }
  #if DEBUG_INIT_CENTROIDS
    cout << endl << "Centroids" << endl; 
    for (i = 0; i < k; ++i) 
      cout << *(centroids[i]) << endl;
  #endif

  CHECK_CUDA_ERROR(hipHostAlloc(&h_centroids, CENTROIDS_BYTES, hipHostMallocDefault));
  for (size_t i = 0; i < k; ++i) {
    for (size_t j = 0; j < d; ++j) {
      h_centroids[i * d + j] = centroids[i]->get(j);
    }
  }
  CHECK_CUDA_ERROR(hipMalloc(&d_centroids, CENTROIDS_BYTES));
}

Kmeans::Kmeans (size_t _n, unsigned int _d, unsigned int _k, Point<DATA_TYPE>** _points, hipDeviceProp_t* _deviceProps)
    : n(_n), d(_d), k(_k),
    POINTS_BYTES(_n * _d * sizeof(DATA_TYPE)),
    CENTROIDS_BYTES(_k * _d * sizeof(DATA_TYPE)),
    points(_points),
    deviceProps(_deviceProps) {

  CHECK_CUDA_ERROR(hipHostAlloc(&h_points, POINTS_BYTES, hipHostMallocDefault));
  for (size_t i = 0; i < n; ++i) {
    for (size_t j = 0; j < d; ++j) {
      h_points[i * d + j] = _points[i]->get(j);
    }
  }
  CHECK_CUDA_ERROR(hipMalloc(&d_points, POINTS_BYTES));
  CHECK_CUDA_ERROR(hipMemcpy(d_points, h_points, POINTS_BYTES, hipMemcpyHostToDevice));

  init_centroids(_points);
}

Kmeans::~Kmeans () {
  CHECK_CUDA_ERROR(hipHostFree(h_points));
  CHECK_CUDA_ERROR(hipHostFree(h_centroids));
  CHECK_CUDA_ERROR(hipHostFree(h_last_centroids));
  CHECK_CUDA_ERROR(hipHostFree(h_points_clusters));
  CHECK_CUDA_ERROR(hipFree(d_centroids));
  CHECK_CUDA_ERROR(hipFree(d_points));
}

uint64_t Kmeans::run (uint64_t maxiter) {
  uint64_t converged = maxiter;

  /* INIT */
  DATA_TYPE* d_distances;
  CHECK_CUDA_ERROR(hipMalloc(&d_distances, n * k * sizeof(DATA_TYPE)));
  uint32_t* d_points_clusters;
  CHECK_CUDA_ERROR(hipMalloc(&d_points_clusters, n * sizeof(uint32_t)));
  CHECK_CUDA_ERROR(hipHostMalloc(&h_points_clusters, n * sizeof(uint32_t)));
  uint32_t* d_clusters_len;
  CHECK_CUDA_ERROR(hipMalloc(&d_clusters_len, k * sizeof(uint32_t)));

  uint64_t iter = 0;

  #if COMPUTE_DISTANCES_KERNEL == 1
    const uint32_t dist_max_points_per_warp = deviceProps->warpSize / d;
    dim3 dist_grid_dim(ceil(((float) n) / dist_max_points_per_warp), k);
    dim3 dist_block_dim(dist_max_points_per_warp * d);
    uint32_t dist_kernel_sh_mem = k * dist_max_points_per_warp * d * sizeof(DATA_TYPE);
  #elif COMPUTE_DISTANCES_KERNEL == 2
    const uint32_t dist_max_points_per_warp = deviceProps->warpSize / next_pow_2(d); // FIXME k > 32
    dim3 dist_grid_dim(ceil(((float) n) / dist_max_points_per_warp), k);
    dim3 dist_block_dim(dist_max_points_per_warp * next_pow_2(d));
    uint32_t dist_kernel_sh_mem = 0;
  #else
    dim3 dist_grid_dim(n, k);
    dim3 dist_block_dim(d);
    uint32_t dist_kernel_sh_mem = 0;
  #endif

  #if ARGMIN_KERNEL == 1
    dim3 argmin_grid_dim(n);
    dim3 argmin_block_dim(max(next_pow_2(k), deviceProps->warpSize));
    uint32_t argmin_warps_per_block = (k + deviceProps->warpSize - 1) / deviceProps->warpSize; // Ceil
    uint32_t argmin_kernel_sh_mem = argmin_warps_per_block * sizeof(Pair);
  #endif
  
  dim3 cent_grid_dim(k);
  dim3 cent_block_dim((((int) n) > deviceProps->warpSize) ? next_pow_2((n + 1) / 2) : deviceProps->warpSize, d); 
  int cent_threads_tot = cent_block_dim.x * cent_block_dim.y;
  while (cent_threads_tot > deviceProps->maxThreadsPerBlock) {
    cent_block_dim.x /= 2;
    cent_grid_dim.y *= 2;
    cent_threads_tot = cent_block_dim.x * cent_block_dim.y;
  }  
  size_t cent_sh_mem = 0;
  #if COMPUTE_CENTROIDS_KERNEL == 1
    cent_sh_mem = (cent_block_dim.x / deviceProps->warpSize) * k * d * sizeof(DATA_TYPE);
  #endif

  /* MAIN LOOP */
  while (iter++ < maxiter) {
    /* COMPUTE DISTANCES */
    CHECK_CUDA_ERROR(hipMemcpy(d_centroids, h_centroids, CENTROIDS_BYTES, hipMemcpyHostToDevice));
    if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u), Sh.mem. %uB\n", "compute_distances", dist_grid_dim.x, dist_grid_dim.y, dist_grid_dim.z, dist_block_dim.x, dist_block_dim.y, dist_block_dim.z, dist_kernel_sh_mem);
    #if PERFORMANCES_KERNEL_DISTANCES
      hipEvent_t e_perf_dist_start, e_perf_dist_stop;
      hipEventCreate(&e_perf_dist_start);
      hipEventCreate(&e_perf_dist_stop);
      hipEventRecord(e_perf_dist_start);
    #endif
    #if COMPUTE_DISTANCES_KERNEL == 1
      compute_distances_shmem<<<dist_grid_dim, dist_block_dim, dist_kernel_sh_mem>>>(d_distances, d_centroids, d_points, dist_max_points_per_warp, d);
    #elif COMPUTE_DISTANCES_KERNEL == 2     
      compute_distances_shfl<<<dist_grid_dim, dist_block_dim>>>(d_distances, d_centroids, d_points, n, dist_max_points_per_warp, d, next_pow_2(d));
    #else
      compute_distances_one_point_per_warp<<<dist_grid_dim, dist_block_dim>>>(d_distances, d_centroids, d_points);
    #endif
    CHECK_CUDA_ERROR(hipDeviceSynchronize());  
    #if PERFORMANCES_KERNEL_DISTANCES
      hipEventRecord(e_perf_dist_stop);
      hipEventSynchronize(e_perf_dist_stop);
      float e_perf_dist_ms = 0;
      hipEventElapsedTime(&e_perf_dist_ms, e_perf_dist_start, e_perf_dist_stop);
      printf(CYAN "[PERFORMANCE]" RESET " compute_distances time: %.8f\n", e_perf_dist_ms / 1000);
      hipEventDestroy(e_perf_dist_start);
      hipEventDestroy(e_perf_dist_stop);
    #endif

    #if DEBUG_KERNEL_DISTANCES
      printf(GREEN "[DEBUG_KERNEL_DISTANCES]\n");
      DATA_TYPE tmp[n * k];
      CHECK_CUDA_ERROR(hipMemcpy(tmp, d_distances, n * k * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
      hipDeviceSynchronize();
      for (uint32_t i = 0; i < n; ++i)
        for (uint32_t j = 0; j < k; ++j)
          printf("%-2u %-2u -> %.3f\n", i, j, tmp[i * k + j]);
      cout << RESET << endl;
    #endif


    /* ASSIGN POINTS TO NEW CLUSTERS */
    #if DEBUG_KERNEL_ARGMIN && ARGMIN_KERNEL == 0
      printf(GREEN "[DEBUG_KERNEL_ARGMIN]\n" RESET);
    #endif
    #if PERFORMANCES_KERNEL_ARGMIN
      hipEvent_t e_perf_argmin_start, e_perf_argmin_stop;
      hipEventCreate(&e_perf_argmin_start);
      hipEventCreate(&e_perf_argmin_stop);
      hipEventRecord(e_perf_argmin_start);
    #endif
    #if ARGMIN_KERNEL == 0
      memset(h_clusters_len, 0, k * sizeof(uint64_t));
      for (size_t i = 0; i < n; i++) {
        hipcub::KeyValuePair<int32_t, DATA_TYPE> *d_argmin = NULL;
        CHECK_CUDA_ERROR(hipMalloc(&d_argmin, sizeof(int32_t) + sizeof(DATA_TYPE)));
        // Allocate temporary storage
        void *d_temp_storage = NULL; size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_distances, d_argmin, k);
        CHECK_CUDA_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));
        
        // Run argmin-reduction
        hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_distances + i * k, d_argmin, k);

        int32_t argmin_idx;
        DATA_TYPE argmin_val;
        CHECK_CUDA_ERROR(hipMemcpy(&argmin_idx, &(d_argmin->key), sizeof(int32_t), hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipMemcpy(&argmin_val, &(d_argmin->value), sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipFree(d_temp_storage));
        CHECK_CUDA_ERROR(hipFree(d_argmin));
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        #if DEBUG_KERNEL_ARGMIN
          printf(GREEN "Argmin point %lu: %d %.3f" RESET "\n", i, argmin_idx, argmin_val);
        #endif

        ++h_clusters_len[argmin_idx];
        h_points_clusters[i] = argmin_idx;
      }
      CHECK_CUDA_ERROR(hipMemcpy(d_points_clusters, h_points_clusters, n * sizeof(uint32_t), hipMemcpyHostToDevice));
    #else
      CHECK_CUDA_ERROR(hipMemset(d_clusters_len, 0, k * sizeof(uint32_t)));
      DATA_TYPE infty = numeric_limits<DATA_TYPE>::infinity();
      if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u), Sh.mem. %uB\n", "clusters_argmin_shfl", argmin_grid_dim.x, argmin_grid_dim.y, argmin_grid_dim.z, argmin_block_dim.x, argmin_block_dim.y, argmin_block_dim.z, argmin_kernel_sh_mem);
      clusters_argmin_shfl<<<argmin_grid_dim, argmin_block_dim, argmin_kernel_sh_mem>>>(n, k, d_distances, d_points_clusters, d_clusters_len, argmin_warps_per_block, infty);
      hipDeviceSynchronize();
    #endif
    #if PERFORMANCES_KERNEL_ARGMIN
      hipEventRecord(e_perf_argmin_stop);
      hipEventSynchronize(e_perf_argmin_stop);
      float e_perf_argmin_ms = 0;
      hipEventElapsedTime(&e_perf_argmin_ms, e_perf_argmin_start, e_perf_argmin_stop);
      printf(CYAN "[PERFORMANCE]" RESET " clusters_argmin_shfl time: %.8f\n", e_perf_argmin_ms / 1000);
      hipEventDestroy(e_perf_argmin_stop);
      hipEventDestroy(e_perf_argmin_start);
    #endif
    #if DEBUG_KERNEL_ARGMIN
      #if ARGMIN_KERNEL == 0
        printf("\n");
      #elif ARGMIN_KERNEL == 1
        printf(GREEN "[DEBUG_KERNEL_ARGMIN]\n" RESET);
        uint32_t tmp1[n];
        CHECK_CUDA_ERROR(hipMemcpy(tmp1, d_points_clusters, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        printf(GREEN "p  -> c\n");
        for (uint32_t i = 0; i < n; ++i)
            printf("%-2u -> %-2u\n", i, tmp1[i]);
        cout << RESET << endl;
      #endif
    #endif

    /* COMPUTE NEW CENTROIDS */
    
    CHECK_CUDA_ERROR(hipMemset(h_centroids, 0, k * d * sizeof(DATA_TYPE)));
    CHECK_CUDA_ERROR(hipMemset(d_centroids, 0, k * d * sizeof(DATA_TYPE)));

    #if PERFORMANCES_KERNEL_CENTROIDS
      hipEvent_t e_perf_cent_start, e_perf_cent_stop;
      hipEventCreate(&e_perf_cent_start);
      hipEventCreate(&e_perf_cent_stop);
      hipEventRecord(e_perf_cent_start);
    #endif

    if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u), Sh.mem. %luB\n", "compute_centroids", cent_grid_dim.x, cent_grid_dim.y, cent_grid_dim.z, cent_block_dim.x, cent_block_dim.y, cent_block_dim.z, cent_sh_mem);
    
    #if COMPUTE_CENTROIDS_KERNEL == 1
      compute_centroids_shfl_shrd<<<cent_grid_dim, cent_block_dim, cent_sh_mem>>>(d_centroids, d_points, d_points_clusters, d_clusters_len, n, d);
    #else 
      compute_centroids_shfl<<<cent_grid_dim, cent_block_dim>>>(d_centroids, d_points, d_points_clusters, d_clusters_len, n, d);
    #endif
    CHECK_CUDA_ERROR(hipDeviceSynchronize());    

    #if PERFORMANCES_KERNEL_CENTROIDS
      hipEventRecord(e_perf_cent_stop);
      hipEventSynchronize(e_perf_cent_stop);
      float e_perf_cent_ms = 0;
      hipEventElapsedTime(&e_perf_cent_ms, e_perf_cent_start, e_perf_cent_stop);
      printf(CYAN "[PERFORMANCE]" RESET " compute_centroids time: %.8f\n", e_perf_cent_ms / 1000);
      hipEventDestroy(e_perf_cent_start);
      hipEventDestroy(e_perf_cent_stop);
    #endif

    #if DEBUG_KERNEL_CENTROIDS
      uint32_t* h_clusters_len;
      CHECK_CUDA_ERROR(hipHostMalloc(&h_clusters_len, k * sizeof(uint32_t)));
      CHECK_CUDA_ERROR(hipMemcpy(h_points_clusters, d_points_clusters, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
      CHECK_CUDA_ERROR(hipMemcpy(h_clusters_len,    d_clusters_len,    k * sizeof(uint32_t), hipMemcpyDeviceToHost));
      for (uint32_t i = 0; i < n; ++i) {
        for (uint32_t j = 0; j < d; ++j) {
          h_centroids[h_points_clusters[i] * d + j] += h_points[i * d + j];
        }
      } 
      for (uint32_t i = 0; i < k; ++i) {
        for (uint32_t j = 0; j < d; ++j) {
          uint64_t count = h_clusters_len[i] > 1 ? h_clusters_len[i] : 1; 
          DATA_TYPE scale = 1.0 / ((double) count); 
          h_centroids[i * d + j] *= scale;    
        }
      } 
      cout << GREEN "[DEBUG_KERNEL_CENTROIDS]" << endl;
      cout << endl << "CENTROIDS (CPU)" << endl;
      for (uint32_t i = 0; i < k; ++i) {
        for (uint32_t j = 0; j < d; ++j)
          printf("%.3f, ", h_centroids[i * d + j]);
        cout << endl;
      }
      CHECK_CUDA_ERROR(hipMemset(h_centroids, 0, d * k * sizeof(DATA_TYPE)));     
      CHECK_CUDA_ERROR(hipMemcpy(h_centroids, d_centroids, d * k * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
      cout << endl << "CENTROIDS (GPU)" << endl;
      for (uint32_t i = 0; i < k; ++i) {
        for (uint32_t j = 0; j < d; ++j)
          printf("%.3f, ", h_centroids[i * d + j]);
        cout << endl;
      }
      cout << RESET << endl;
      CHECK_CUDA_ERROR(hipHostFree(h_clusters_len));
    #endif

    CHECK_CUDA_ERROR(hipMemcpy(h_centroids, d_centroids, d * k * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
    
    /* CHECK IF CONVERGED */
    if (iter > 1 && cmp_centroids()) { converged = iter; break; } // Exit
    else { memcpy(h_last_centroids, h_centroids, CENTROIDS_BYTES); } // Copy current centroids
  }
  /* MAIN LOOP END */

  /* COPY BACK RESULTS*/
  CHECK_CUDA_ERROR(hipMemcpy(h_points_clusters, d_points_clusters, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
  for (size_t i = 0; i < n; i++) {
    points[i]->setCluster(h_points_clusters[i]);
  }
  
  /* FREE MEMORY */
  CHECK_CUDA_ERROR(hipFree(d_distances));
  CHECK_CUDA_ERROR(hipFree(d_points_clusters));
  CHECK_CUDA_ERROR(hipFree(d_clusters_len));

  return converged;
}

bool Kmeans::cmp_centroids () {
  const DATA_TYPE EPSILON = numeric_limits<DATA_TYPE>::epsilon();
  DATA_TYPE dist_sum = 0, norm = 0;
  
  for (size_t i = 0; i < k; ++i) {
    for (size_t j = 0; j < d; ++j) {
      DATA_TYPE dist = fabs(h_centroids[i * d + j] - h_last_centroids[i * d + j]);
      dist_sum += dist * dist;
      norm += h_last_centroids[i * d + j] * h_last_centroids[i * d + j];
    }
    if (sqrt(dist_sum) > EPSILON) { return false; }
  }

  return true;
}

void Kmeans::to_csv(ostream& o, char separator) {
  o << "cluster" << separator;
  for (size_t i = 0; i < d; ++i) {
    o << "d" << i;
    if (i != (d - 1)) o << separator;
  }
  o << endl;
  for (size_t i = 0; i < n; ++i) {
    o << h_points_clusters[i] << separator;
    for (size_t j = 0; j < d; ++j) {
      o << setprecision(8) << h_points[i * d + j];
      if (j != (d - 1)) o << separator;
    }
    o << endl;
  }
}