#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <iomanip>
#include <cmath>
#include <limits>
#include <hipblas.h>

#include "include/common.h"
#include "include/colors.h"

#include "cuda_utils.cuh"
#include "kmeans.cuh"

#include "kernels/kernels.cuh"

using namespace std;

const DATA_TYPE INFNTY = numeric_limits<DATA_TYPE>::infinity();

Kmeans::Kmeans (const size_t _n, const uint32_t _d, const uint32_t _k, const float _tol, const int* seed, Point<DATA_TYPE>** _points, hipDeviceProp_t* _deviceProps)
		: n(_n), d(_d), k(_k), tol(_tol),
		POINTS_BYTES(_n * _d * sizeof(DATA_TYPE)),
		CENTROIDS_BYTES(_k * _d * sizeof(DATA_TYPE)),
		points(_points),
		deviceProps(_deviceProps) {

	if (seed) {
		seed_seq s{*seed};
		generator = new mt19937(s);
	}
	else {
		random_device rd;
		generator = new mt19937(rd());
	}

	CHECK_CUDA_ERROR(hipHostAlloc(&h_points, POINTS_BYTES, hipHostMallocDefault));
	for (size_t i = 0; i < n; ++i) {
		for (size_t j = 0; j < d; ++j) {
			h_points[i * d + j] = _points[i]->get(j);
		}
	}
	CHECK_CUDA_ERROR(hipMalloc(&d_points, POINTS_BYTES));
	CHECK_CUDA_ERROR(hipMemcpy(d_points, h_points, POINTS_BYTES, hipMemcpyHostToDevice));

	init_centroids(_points);
	CHECK_CUDA_ERROR(hipMemcpy(d_centroids, h_centroids, d * k * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
}

Kmeans::~Kmeans () {
	delete generator;
	CHECK_CUDA_ERROR(hipHostFree(h_points));
	CHECK_CUDA_ERROR(hipHostFree(h_centroids));
	CHECK_CUDA_ERROR(hipHostFree(h_last_centroids));
	CHECK_CUDA_ERROR(hipHostFree(h_points_clusters));
	CHECK_CUDA_ERROR(hipFree(d_centroids));
	CHECK_CUDA_ERROR(hipFree(d_points));
	if (h_centroids_matrix != NULL) {
		CHECK_CUDA_ERROR(hipHostFree(h_centroids_matrix));
	}
	compute_gemm_distances_free();
}

void Kmeans::init_centroids (Point<DATA_TYPE>** points) {
	uniform_int_distribution<int> random_int(0, n - 1);

	if (COMPUTE_DISTANCES_KERNEL == 2) {
		CENTROIDS_BYTES += (k * sizeof(DATA_TYPE)); // Be aware
		CHECK_CUDA_ERROR(hipHostAlloc(&h_centroids_matrix, CENTROIDS_BYTES, hipHostMallocDefault));
	} else {
		h_centroids_matrix = NULL;
	}

	CHECK_CUDA_ERROR(hipHostAlloc(&h_centroids, CENTROIDS_BYTES, hipHostMallocDefault));
	CHECK_CUDA_ERROR(hipHostAlloc(&h_last_centroids, CENTROIDS_BYTES, hipHostMallocDefault));

	unsigned int i = 0;
	vector<Point<DATA_TYPE>*> usedPoints;
	Point<DATA_TYPE>* centroids[k];
	while (i < k) {
		Point<DATA_TYPE>* p = points[random_int(*generator)];
		bool found = false;
		for (auto p1 : usedPoints) {
			if ((*p1) == (*p)) {
				found = true;
				break;
			}
		}
		if (!found) {
			centroids[i] = new Point<DATA_TYPE>(p);
			usedPoints.push_back(p);
			++i;
		}
	}

#if DEBUG_INIT_CENTROIDS
    cout << endl << "Centroids" << endl;
    for (i = 0; i < k; ++i)
        cout << *(centroids[i]) << endl;
#endif

	for (size_t i = 0; i < k; ++i) {
		for (size_t j = 0; j < d; ++j) {
			h_centroids[i * d + j] = centroids[i]->get(j); // Row major
#if COMPUTE_DISTANCES_KERNEL==2
            h_centroids_matrix[(j + 1) * k + i] = centroids[i]->get(j); // Col major
#endif
		}
	}

#if COMPUTE_DISTANCES_KERNEL==2
    for (size_t i = 0; i < k; ++i)
        h_centroids_matrix[i] = 1; // Static prefix
#endif

	memcpy(h_last_centroids, h_centroids, CENTROIDS_BYTES);
	CHECK_CUDA_ERROR(hipMalloc(&d_centroids, CENTROIDS_BYTES));
}

uint64_t Kmeans::run (uint64_t maxiter) {
    uint64_t converged = maxiter;

    /* INIT */
    DATA_TYPE* d_distances;
    CHECK_CUDA_ERROR(hipMalloc(&d_distances, n * k * sizeof(DATA_TYPE)));

    uint32_t* d_points_clusters;
    CHECK_CUDA_ERROR(hipMalloc(&d_points_clusters, n * sizeof(uint32_t)));
    CHECK_CUDA_ERROR(hipHostMalloc(&h_points_clusters, n * sizeof(uint32_t)));

    uint32_t* d_clusters_len;
    CHECK_CUDA_ERROR(hipMalloc(&d_clusters_len, k * sizeof(uint32_t)));

    uint64_t iter = 0;
    const uint32_t rounds = ((d - 1) / deviceProps->warpSize) + 1;

#if COMPUTE_DISTANCES_KERNEL==1
    dim3 dist_grid_dim, dist_block_dim;
    uint32_t dist_max_points_per_warp;
    schedule_distances_kernel(deviceProps, n, d, k, 
                                &dist_grid_dim, &dist_block_dim, 
                                &dist_max_points_per_warp);
#elif COMPUTE_DISTANCES_KERNEL==2

    DATA_TYPE* d_points_assoc_matrices;
    DATA_TYPE* d_centroids_matrix;

    uint32_t d1 = d + 1;
    uint32_t nd1d1 = n * d1 * d1;

    // Associated to POINTS (centers change after every iteration)
    CHECK_CUDA_ERROR(hipMalloc(&d_points_assoc_matrices, nd1d1 * sizeof(DATA_TYPE)));
    CHECK_CUDA_ERROR(hipMemset(d_points_assoc_matrices, 0, nd1d1 * sizeof(DATA_TYPE)));
    CHECK_CUDA_ERROR(hipMalloc(&d_centroids_matrix, CENTROIDS_BYTES));

    dim3 dist_assoc_matrices_grid_dim(n);
    dim3 dist_assoc_matrices_block_dim(min(next_pow_2(d), deviceProps->warpSize));

#if DEBUG_KERNELS_INVOKATION
    printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u), Sh.mem. %uB\n", "compute_point_associated_matrices", dist_assoc_matrices_grid_dim.x, dist_assoc_matrices_grid_dim.y, dist_assoc_matrices_grid_dim.z, dist_assoc_matrices_block_dim.x, dist_assoc_matrices_block_dim.y, dist_assoc_matrices_block_dim.z, 0);
#endif

    for (uint32_t i = 0; i < rounds; i++) {
        compute_point_associated_matrices<<<dist_assoc_matrices_grid_dim, 
                                            dist_assoc_matrices_block_dim>>>
                                            (d_points, d_points_assoc_matrices, d, i);
    }

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS_ERROR(hipblasCreate(&cublasHandle));

#elif COMPUTE_DISTANCES_KERNEL==3

    /* Initialize P and C using d_points and d_centroids */
    
    DATA_TYPE * d_P;
    DATA_TYPE * d_C;

    size_t p_rows = n;
    size_t p_cols = 3*d;
    size_t p_size = p_rows*p_cols;

    size_t c_rows = 3*d;
    size_t c_cols = k;
    size_t c_size = c_rows*c_cols;

    CHECK_CUDA_ERROR(hipMalloc(&d_P, sizeof(DATA_TYPE)*p_rows*p_cols));

    uint32_t p_mat_block_dim(min((size_t)deviceProps->maxThreadsPerBlock, p_rows));
    uint32_t p_mat_grid_dim(p_cols);
    uint32_t p_rounds = ceil((float)p_rows / (float)p_mat_block_dim);

    compute_p_matrix<<<p_mat_grid_dim, p_mat_block_dim>>>(d_points, d_P, d, n, k, p_rounds);

    //Debug
    //DATA_TYPE * h_points_debug = new DATA_TYPE[n*d];
    //CHECK_CUDA_ERROR(hipMemcpy(h_points_debug, d_points, sizeof(DATA_TYPE)*n*d, hipMemcpyDeviceToHost));
    //cout<<"Points matrix"<<endl;
    //printMatrixRowMaj(h_points_debug, n, d);

    //DATA_TYPE * h_P_debug = new DATA_TYPE[p_size];
    //CHECK_CUDA_ERROR(hipMemcpy(h_P_debug, d_P, sizeof(DATA_TYPE)*p_size, hipMemcpyDeviceToHost));
    //cout<<"Computed P matrix"<<endl;
    //printMatrixColMaj(h_P_debug, p_rows, p_cols);

    //check_p_correctness(h_P_debug, h_points_debug, n, d);
    
    //delete[] h_P_debug;
    //delete[] h_points_debug;

    // Malloc C here, but don't initialize it yet because we need to do that once per iteration
    CHECK_CUDA_ERROR(hipMalloc(&d_C, sizeof(DATA_TYPE)*c_rows*c_cols));

#endif

    dim3 argmin_grid_dim, argmin_block_dim;
    uint32_t argmin_warps_per_block, argmin_sh_mem;
    schedule_argmin_kernel(deviceProps, n, k, 
                            &argmin_grid_dim, &argmin_block_dim, 
                            &argmin_warps_per_block, &argmin_sh_mem);

    dim3 cent_grid_dim, cent_block_dim;
    schedule_centroids_kernel(deviceProps, n, d, k, &cent_grid_dim, &cent_block_dim);

    /* MAIN LOOP */
    while (iter++ < maxiter) {
    /* COMPUTE DISTANCES */

#if COMPUTE_DISTANCES_KERNEL==1
        if (DEBUG_KERNELS_INVOKATION) 
            printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u), Sh.mem. %uB\n", "compute_distances", dist_grid_dim.x, dist_grid_dim.y, dist_grid_dim.z, dist_block_dim.x, dist_block_dim.y, dist_block_dim.z, 0);
#elif COMPUTE_DISTANCES_KERNEL==2
        if (DEBUG_KERNELS_INVOKATION) 
            printf(YELLOW "[KERNEL]" RESET " Matmul\n");
#endif

#if PERFORMANCES_KERNEL_DISTANCES

        hipEvent_t e_perf_dist_start, e_perf_dist_stop;

        hipEventCreate(&e_perf_dist_start);
        hipEventCreate(&e_perf_dist_stop);
        hipEventRecord(e_perf_dist_start);

#endif

#if COMPUTE_DISTANCES_KERNEL==1

        if (static_cast<int>(d)<=deviceProps->warpSize) {
            compute_distances_shfl<<<dist_grid_dim, 
                                    dist_block_dim>>>
                                    (d_distances, d_centroids, d_points, 
                                     n, dist_max_points_per_warp, d, 
                                     log2(next_pow_2(d)) > 0 ? log2(next_pow_2(d)) : 1);
        } else {
            for (uint32_t i = 0; i < rounds; i++) {
                compute_distances_one_point_per_warp<<<dist_grid_dim, 
                                                        dist_block_dim>>>
                                                        (d_distances, d_centroids, 
                                                         d_points, d, next_pow_2(d), i);
            }
        }

#elif COMPUTE_DISTANCES_KERNEL==2
        CHECK_CUBLAS_ERROR(hipblasSetMatrix(k, d1, sizeof(DATA_TYPE), 
                                            h_centroids_matrix, k, 
                                            d_centroids_matrix, k)); 

        compute_gemm_distances(cublasHandle, deviceProps, 
                                d1, n, k, 
                                d_points_assoc_matrices, d_centroids_matrix, 
                                d_distances);

#elif COMPUTE_DISTANCES_KERNEL==3
       
        uint32_t compute_c_grid_dim = c_cols;
        uint32_t compute_c_block_dim = min((size_t)deviceProps->maxThreadsPerBlock, c_rows/3);
        uint32_t c_rounds = ceil((float)c_rows / (float)compute_c_block_dim);
        compute_c_matrix<<<compute_c_grid_dim, compute_c_block_dim>>>(d_centroids, d_C, d, n, k, c_rounds); 

        //cout<<"Centroids"<<endl;
        //printMatrixRowMaj(h_centroids, k, d);

        //DATA_TYPE * h_C_debug = new DATA_TYPE[k*d];
        //CHECK_CUDA_ERROR(hipMemcpy(h_C_debug, d_C, sizeof(DATA_TYPE)*c_size, hipMemcpyDeviceToHost));
        //cout<<"C matrix"<<endl;
        //printMatrixColMaj(h_C_debug, c_rows, c_cols);

        //check_c_correctness(h_C_debug, h_centroids, k, d);

        //delete[] h_C_debug;

#endif

#if PERFORMANCES_KERNEL_DISTANCES

        hipEventRecord(e_perf_dist_stop);
        hipEventSynchronize(e_perf_dist_stop);

        float e_perf_dist_ms = 0;
        hipEventElapsedTime(&e_perf_dist_ms, e_perf_dist_start, e_perf_dist_stop);
        printf(CYAN "[PERFORMANCE]" RESET " compute_distances time: %.8f\n", e_perf_dist_ms / 1000);

        hipEventDestroy(e_perf_dist_start);
        hipEventDestroy(e_perf_dist_stop);

#endif

#if DEBUG_KERNEL_DISTANCES

        printf(GREEN "[DEBUG_KERNEL_DISTANCES]\n");

#if COMPUTE_DISTANCES_KERNEL==2

        cout << "Centroids matrix" << endl;
        printMatrixColMaj(h_centroids_matrix, k, d1);
        cout << endl;

        DATA_TYPE tmp_assoc_mat[(d + 1) * (d + 1)];

        uint32_t d1d1 = d1 * d1;

        for (size_t i = 0; i < 1; i++) {
            cout << "Point " << i << " associated matrix" << endl;
            CHECK_CUDA_ERROR(hipMemcpy(tmp_assoc_mat, d_points_assoc_matrices + (d1d1 * i), d1d1 * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
            printMatrixColMaj(tmp_assoc_mat, d1, d1);
            cout << endl;
        }

#endif

        DATA_TYPE* cpu_distances = new DATA_TYPE[n * k];

        for (uint32_t ni = 0; ni < n; ++ni) {
            for (uint32_t ki = 0; ki < k; ++ki) {
                DATA_TYPE dist = 0, tmp;
                for (uint32_t di = 0; di < d; ++di) {
                    tmp = h_points[ni * d + di] - h_centroids[ki * d + di];
                    dist += tmp * tmp;
                }
                cpu_distances[ni * k + ki] = dist;
            }
        }

        DATA_TYPE* tmp_dist = new DATA_TYPE[n * k];

        int anyError = 0;

        CHECK_CUDA_ERROR(hipMemcpy(tmp_dist, d_distances, 
                                    n * k * sizeof(DATA_TYPE), 
                                    hipMemcpyDeviceToHost));

        for (uint32_t i = 0; i < n; ++i)
            for (uint32_t j = 0; j < k; ++j)
                if (fabs(tmp_dist[i * k + j] - cpu_distances[i * k + j]) > 0.001) {
                    printf("N=%-2u K=%-2u -> GPU=%.4f CPU=%.4f diff: %.8f\n", 
                            i, j, tmp_dist[i * k + j], cpu_distances[i * k + j], 
                            fabs(tmp_dist[i * k + j] - cpu_distances[i * k + j]));
                    anyError = 1;
                }
        cout << (anyError ? "Something wrong" : "Everything alright") << RESET << endl;
        delete[] cpu_distances;
        delete[] tmp_dist;
#endif

		////////////////////////////////////////* ASSIGN POINTS TO NEW CLUSTERS */////////////////////////////////////////

#if PERFORMANCES_KERNEL_ARGMIN

        hipEvent_t e_perf_argmin_start, e_perf_argmin_stop;

        hipEventCreate(&e_perf_argmin_start);
        hipEventCreate(&e_perf_argmin_stop);
        hipEventRecord(e_perf_argmin_start);

#endif

		if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u), Sh.mem. %uB\n", "clusters_argmin_shfl", argmin_grid_dim.x, argmin_grid_dim.y, argmin_grid_dim.z, argmin_block_dim.x, argmin_block_dim.y, argmin_block_dim.z, argmin_sh_mem);

		CHECK_CUDA_ERROR(hipMemset(d_clusters_len, 0, k * sizeof(uint32_t)));
		clusters_argmin_shfl<<<argmin_grid_dim, 
                                argmin_block_dim, 
                                argmin_sh_mem>>>
                                (n, k, d_distances, d_points_clusters, 
                                 d_clusters_len, argmin_warps_per_block, INFNTY);

#if PERFORMANCES_KERNEL_ARGMIN

        hipEventRecord(e_perf_argmin_stop);
        hipEventSynchronize(e_perf_argmin_stop);

        float e_perf_argmin_ms = 0;
        hipEventElapsedTime(&e_perf_argmin_ms, e_perf_argmin_start, e_perf_argmin_stop);

        printf(CYAN "[PERFORMANCE]" RESET " clusters_argmin_shfl time: %.8f\n", e_perf_argmin_ms / 1000);

        hipEventDestroy(e_perf_argmin_stop);
        hipEventDestroy(e_perf_argmin_start);

#endif

#if DEBUG_KERNEL_ARGMIN

        printf(GREEN "[DEBUG_KERNEL_ARGMIN]\n" RESET);

        uint32_t tmp1[n];
        CHECK_CUDA_ERROR(hipMemcpy(tmp1, 
                                    d_points_clusters, n * sizeof(uint32_t), 
                                    hipMemcpyDeviceToHost));

        printf(GREEN "p  -> c\n");
        for (uint32_t i = 0; i < n; ++i)
                printf("%-2u -> %-2u\n", i, tmp1[i]);
        cout << RESET << endl;

#endif

		///////////////////////////////////////////* COMPUTE NEW CENTROIDS *///////////////////////////////////////////

		CHECK_CUDA_ERROR(hipMemset(d_centroids, 0, k * d * sizeof(DATA_TYPE)));

#if PERFORMANCES_KERNEL_CENTROIDS

        hipEvent_t e_perf_cent_start, e_perf_cent_stop;

        hipEventCreate(&e_perf_cent_start);
        hipEventCreate(&e_perf_cent_stop);
        hipEventRecord(e_perf_cent_start);

#endif

		if (DEBUG_KERNELS_INVOKATION) 
            printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u)\n", "compute_centroids", cent_grid_dim.x, cent_grid_dim.y, cent_grid_dim.z, cent_block_dim.x, cent_block_dim.y, cent_block_dim.z);

		for (uint32_t i = 0; i < rounds; i++) {
			compute_centroids_shfl<<<cent_grid_dim, 
                                    cent_block_dim>>>
                                    (d_centroids, d_points, 
                                     d_points_clusters, d_clusters_len, 
                                     n, d, k, i);
		}

#if PERFORMANCES_KERNEL_CENTROIDS

        hipEventRecord(e_perf_cent_stop);
        hipEventSynchronize(e_perf_cent_stop);

        float e_perf_cent_ms = 0;
        hipEventElapsedTime(&e_perf_cent_ms, e_perf_cent_start, e_perf_cent_stop);

        printf(CYAN "[PERFORMANCE]" RESET " compute_centroids time: %.8f\n", e_perf_cent_ms / 1000);

        hipEventDestroy(e_perf_cent_start);
        hipEventDestroy(e_perf_cent_stop);
#endif

#if DEBUG_KERNEL_CENTROIDS

        CHECK_CUDA_ERROR(hipMemset(h_centroids, 0, k * d * sizeof(DATA_TYPE)));
        CHECK_CUDA_ERROR(hipMemcpy(h_points_clusters, d_points_clusters, n * sizeof(uint32_t), hipMemcpyDeviceToHost));

        uint32_t* h_clusters_len;
        CHECK_CUDA_ERROR(hipHostMalloc(&h_clusters_len, k * sizeof(uint32_t)));
        CHECK_CUDA_ERROR(hipMemcpy(h_clusters_len, 
                                    d_clusters_len,	
                                    k * sizeof(uint32_t), 
                                    hipMemcpyDeviceToHost));

        for (uint32_t i = 0; i < n; ++i) {
            for (uint32_t j = 0; j < d; ++j) {
                h_centroids[h_points_clusters[i] * d + j] += h_points[i * d + j];
            }
        }

        for (uint32_t i = 0; i < k; ++i) {
            for (uint32_t j = 0; j < d; ++j) {
                uint64_t count = h_clusters_len[i] > 1 ? h_clusters_len[i] : 1;
                DATA_TYPE scale = 1.0 / ((double) count);
                h_centroids[i * d + j] *= scale;
            }
        }

        cout << GREEN "[DEBUG_KERNEL_CENTROIDS]" << endl;
        cout << endl << "CENTROIDS (CPU)" << endl;
        for (uint32_t i = 0; i < k; ++i) {
            for (uint32_t j = 0; j < d; ++j)
                printf("%.3f, ", h_centroids[i * d + j]);
            cout << endl;
        }

        CHECK_CUDA_ERROR(hipMemset(h_centroids, 0, d * k * sizeof(DATA_TYPE)));
        CHECK_CUDA_ERROR(hipMemcpy(h_centroids, d_centroids, 
                                    d * k * sizeof(DATA_TYPE), 
                                    hipMemcpyDeviceToHost));

        cout << endl << "CENTROIDS (GPU)" << endl;
        for (uint32_t i = 0; i < k; ++i) {
            for (uint32_t j = 0; j < d; ++j)
                printf("%.3f, ", h_centroids[i * d + j]);
            cout << endl;
        }

        cout << RESET << endl;
        CHECK_CUDA_ERROR(hipHostFree(h_clusters_len));

#endif

		CHECK_CUDA_ERROR(hipMemcpy(h_centroids, 
                                    d_centroids, 
                                    d * k * sizeof(DATA_TYPE), 
                                    hipMemcpyDeviceToHost));
		CHECK_CUDA_ERROR(hipDeviceSynchronize());

		/////////////////////////////////////////////* CHECK IF CONVERGED */////////////////////////////////////////////

		// Check exit
		if (iter > 1 && cmp_centroids()) {
			converged = iter;
			break;
		}

		// Copy current centroids
		memcpy(h_last_centroids, h_centroids, CENTROIDS_BYTES);

#if COMPUTE_DISTANCES_KERNEL==2
        /* UPDATE h_centroids_matrix */
        for (size_t i = 0; i < k; ++i) {
            h_centroids_matrix[i] = 1; // Static prefix
            for (size_t j = 0; j < d; ++j) {
                h_centroids_matrix[IDX2C(i, j + 1, k)] = h_centroids[i * d + j]; // Row maj to Col maj
            }
        }
#endif

	}
	/* MAIN LOOP END */

#if DEBUG_INIT_CENTROIDS
    cout << endl << "Centroids" << endl;
    printMatrixRowMaj(h_centroids, k, d);
#endif

	/* COPY BACK RESULTS*/
	CHECK_CUDA_ERROR(hipMemcpy(h_points_clusters, d_points_clusters, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	for (size_t i = 0; i < n; i++) {
		points[i]->setCluster(h_points_clusters[i]);
	}

	/* FREE MEMORY */
	CHECK_CUDA_ERROR(hipFree(d_distances));
	CHECK_CUDA_ERROR(hipFree(d_points_clusters));
	CHECK_CUDA_ERROR(hipFree(d_clusters_len));

#if COMPUTE_DISTANCES_KERNEL==2

    CHECK_CUDA_ERROR(hipFree(d_points_assoc_matrices));
    CHECK_CUDA_ERROR(hipFree(d_centroids_matrix));
    CHECK_CUBLAS_ERROR(hipblasDestroy(cublasHandle));

#elif COMPUTE_DISTANCES_KERNEL==3

    CHECK_CUDA_ERROR(hipFree(d_C));
    CHECK_CUDA_ERROR(hipFree(d_P));

#endif

	return converged;
}

bool Kmeans::cmp_centroids () {
	for (size_t i = 0; i < k; ++i) {
		DATA_TYPE dist_sum = 0;
		for (size_t j = 0; j < d; ++j) {
			DATA_TYPE dist = h_centroids[i * d + j] - h_last_centroids[i * d + j];
			dist_sum += dist * dist;
		}
		if (sqrt(dist_sum) > tol) { return false; }
	}

	return true;
}
