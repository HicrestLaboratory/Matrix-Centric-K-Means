#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <random>
#include <iomanip>
#include <hipcub/hipcub.hpp>
#include <cmath>

#include "../include/common.h"
#include "../include/colors.h"
#include "kmeans.cuh"
#include "../lib/cuda/utils.cuh"

using namespace std;

random_device rd;
seed_seq seed{0}; // FIXME use rd()
mt19937 rng(seed);

__host__ __device__ unsigned int next_pow_2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

#define SHFL_MASK 0xffffffff
/* Device kernels */
__global__ void compute_distances_one_point_per_warp(DATA_TYPE* distances, DATA_TYPE* centers, DATA_TYPE* points) {
  const uint64_t point_offset = blockIdx.x * blockDim.x + threadIdx.x;
  const uint64_t center_offset = blockIdx.y * blockDim.x + threadIdx.x;
  DATA_TYPE dist = points[point_offset] - centers[center_offset];
  dist *= dist;
  
  for (int i = next_pow_2(blockDim.x); i > 0; i /= 2)
    dist += __shfl_down_sync(SHFL_MASK, dist, i);

  if (threadIdx.x == 0) {
    distances[(blockIdx.x * gridDim.y) + blockIdx.y] = dist;
  }
}

__global__ void compute_distances_shmem(DATA_TYPE* distances, DATA_TYPE* centers, DATA_TYPE* points, const uint32_t points_per_warp, const uint32_t d) {
  const uint64_t point_i = (blockIdx.x * points_per_warp) + (threadIdx.x / d);
  const uint64_t center_i = blockIdx.y;
  const uint32_t d_i = threadIdx.x % d;
  const uint64_t dists_i = (center_i * blockDim.y * d) + ((point_i % points_per_warp) * d) + d_i;

  extern __shared__ DATA_TYPE dists[];

  if (threadIdx.x < points_per_warp * d) {
    DATA_TYPE dist = fabs(points[point_i * d + d_i] - centers[center_i * d + d_i]);
    dists[dists_i] = dist * dist;
    __syncthreads();
    if (d_i == 0) {
      for (int i = 1; i < d; i++) {
        dists[dists_i] += dists[dists_i + i];
      }
      distances[(point_i * center_i) + point_i] = dists[dists_i];
    }
  }
}

__global__ void compute_distances_shfl(DATA_TYPE* distances, DATA_TYPE* centers, DATA_TYPE* points, const uint32_t points_n, const uint32_t points_per_warp, const uint32_t d, const uint32_t d_closest_2_pow) {
  const uint64_t point_i = (blockIdx.x * points_per_warp) + (threadIdx.x / d_closest_2_pow);
  const uint64_t center_i = blockIdx.y;
  const uint32_t d_i = threadIdx.x % d_closest_2_pow;

  if (point_i < points_n && d_i < d) {
    DATA_TYPE dist = fabs(points[point_i * d + d_i] - centers[center_i * d + d_i]);
    dist *= dist;
    for (int i = d_closest_2_pow / 2; i > 0; i /= 2) {
      dist += __shfl_down_sync(SHFL_MASK, dist, i);
      // if (point_i == 3) printf("%d  p: %lu c: %lu d: %u v: %.3f\n", i, point_i, center_i, d_i, dist);
    }
    if (d_i == 0) {
      distances[(point_i * gridDim.y) + center_i] = dist;
    }
  }
}

__global__ void compute_centers(DATA_TYPE* centers, DATA_TYPE* points, uint32_t* points_clusters, uint64_t* clusters_len) {
  uint32_t point   = blockIdx.x;
  uint32_t cluster = points_clusters[point];
  uint32_t d       = blockDim.x;
  uint32_t d_i     = threadIdx.x;
  // extern __shared__ DATA_TYPE centers_shared[];
  
  // if (point >= clusters_len[cluster]) { return; }

  DATA_TYPE val = points[point * d + d_i];
  
  // if (cluster == 0 && d_i == 1) printf("cl: %u p: %u d: %u p_d: %.3f\n", cluster, point, d_i, val);
  
  // for (int i = next_pow_2(blockDim.x); i > 0; i /= 2) sum += __shfl_down_sync(SHFL_MASK, sum, i);
  // atomicAdd(centers_shared + cluster * d + d_i, val);
  // __syncthreads();
  
  //if (point == 0) {
  atomicAdd(centers + cluster * d + d_i, val);
  // if (cluster == 0 && d_i == 1) printf("blk: %u %u %u, part_sum: %.3f\n", blockIdx.x, blockIdx.y, threadIdx.x, centers[cluster * d + d_i]);
  //}
}

__global__ void compute_centers_shfl(DATA_TYPE* centers, DATA_TYPE* points, uint32_t* points_clusters, uint64_t n, uint32_t d) {  
  uint32_t cluster_idx = 2 * blockIdx.y * blockDim.x + threadIdx.x;
  uint32_t point_idx   = cluster_idx * blockDim.y + threadIdx.y;
  
  uint32_t cluster_off = blockDim.x;
  uint32_t point_off   = cluster_off * blockDim.y;
  
  float val = 0;

  if (point_idx < n * d && blockIdx.x == points_clusters[cluster_idx]) { 
    val = points[point_idx]; 
  }
  
  if (point_idx + point_off < n * d && blockIdx.x == points_clusters[cluster_idx + cluster_off]) { 
    val += points[point_idx + point_off]; 
  } 

  //if (blockIdx.x == 0 && threadIdx.y == 0) printf("%d %d %d+%d %f\n", blockIdx.x, blockIdx.y, cluster_idx, cluster_idx + cluster_off, val);
  
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    val += __shfl_down_sync(SHFL_MASK, val, offset);
  }
  
  if (threadIdx.x % warpSize == 0) {
    //if (blockIdx.x == 0 && threadIdx.y == 0) { printf("\n%d %d %f\n", threadIdx.x, blockIdx.y, val); }
    atomicAdd(&centers[blockIdx.x * blockDim.y + threadIdx.y], val);
  }
}

/* Kmeans class */
void Kmeans::initCenters (Point<DATA_TYPE>** points) {
  uniform_int_distribution<int> random_int(0, n - 1);
  CHECK_CUDA_ERROR(hipHostAlloc(&h_centers, CENTERS_BYTES, hipHostMallocDefault));
  CHECK_CUDA_ERROR(hipHostAlloc(&h_last_centers, CENTERS_BYTES, hipHostMallocDefault));
  unsigned int i = 0;
  vector<Point<DATA_TYPE>*> usedPoints;
  Point<DATA_TYPE>* centers[k];
  while (i < k) {
    Point<DATA_TYPE>* p = points[random_int(rng)];
    bool found = false;
    for (auto p1 : usedPoints) {
      if ((*p1) == (*p)) { // FIXME Is it better use some min distance??
        found = true;
        break;
      }
    }
    if (!found) {
      for (unsigned int j = 0; j < d; ++j) {
        h_centers[i * d + j] = p->get(j);
      }
      centers[i] = new Point<DATA_TYPE>(p);
      usedPoints.push_back(p);
      ++i;
    }
  }
  if (DEBUG_INIT_CENTERS) { cout << endl << "Centers" << endl; for (i = 0; i < k; ++i) cout << *(centers[i]) << endl; }

  CHECK_CUDA_ERROR(hipHostAlloc(&h_centers, CENTERS_BYTES, hipHostMallocDefault));
  for (size_t i = 0; i < k; ++i) {
    for (size_t j = 0; j < d; ++j) {
      h_centers[i * d + j] = centers[i]->get(j);
    }
  }
  CHECK_CUDA_ERROR(hipMalloc(&d_centers, CENTERS_BYTES));
}

Kmeans::Kmeans (size_t _n, unsigned int _d, unsigned int _k, Point<DATA_TYPE>** _points, hipDeviceProp_t* _deviceProps)
    : n(_n), d(_d), k(_k),
    POINTS_BYTES(_n * _d * sizeof(DATA_TYPE)),
    CENTERS_BYTES(_k * _d * sizeof(DATA_TYPE)),
    points(_points),
    deviceProps(_deviceProps) {

  CHECK_CUDA_ERROR(hipHostAlloc(&h_points, POINTS_BYTES, hipHostMallocDefault));
  for (size_t i = 0; i < n; ++i) {
    for (size_t j = 0; j < d; ++j) {
      h_points[i * d + j] = _points[i]->get(j);
    }
  }
  CHECK_CUDA_ERROR(hipMalloc(&d_points, POINTS_BYTES));
  CHECK_CUDA_ERROR(hipMemcpy(d_points, h_points, POINTS_BYTES, hipMemcpyHostToDevice));

  initCenters(_points);
}

Kmeans::~Kmeans () {
  CHECK_CUDA_ERROR(hipHostFree(h_points));
  CHECK_CUDA_ERROR(hipHostFree(h_centers));
  CHECK_CUDA_ERROR(hipHostFree(h_last_centers));
  CHECK_CUDA_ERROR(hipHostFree(h_points_clusters));
  CHECK_CUDA_ERROR(hipFree(d_centers));
  CHECK_CUDA_ERROR(hipFree(d_points));
}

uint64_t Kmeans::run (uint64_t maxiter) {
  uint64_t converged = maxiter;

  /* INIT */
  DATA_TYPE* d_distances;
  CHECK_CUDA_ERROR(hipMalloc(&d_distances, n * k * sizeof(DATA_TYPE)));
  uint32_t* d_points_clusters;
  CHECK_CUDA_ERROR(hipMalloc(&d_points_clusters, n * sizeof(uint32_t)));
  CHECK_CUDA_ERROR(hipHostMalloc(&h_points_clusters, n * sizeof(uint32_t)));
  uint64_t* h_clusters_len;
  CHECK_CUDA_ERROR(hipHostMalloc(&h_clusters_len, k * sizeof(uint64_t)));
  uint64_t* d_clusters_len;
  CHECK_CUDA_ERROR(hipMalloc(&d_clusters_len, k * sizeof(uint64_t)));

  uint64_t iter = 0;
  uint64_t max_cluster_len = 0;
  dim3 argmin_block_dim(k, d);

  #if COMPUTE_DISTANCES_KERNEL == 1
    const uint32_t dist_max_points_per_warp = deviceProps->warpSize / d;
    dim3 dist_grid_dim(ceil(((float) n) / dist_max_points_per_warp), k);
    dim3 dist_block_dim(dist_max_points_per_warp * d);
    uint32_t dist_kernel_sh_mem = k * dist_max_points_per_warp * d * sizeof(DATA_TYPE);
  #elif COMPUTE_DISTANCES_KERNEL == 2
    const uint32_t dist_max_points_per_warp = deviceProps->warpSize / next_pow_2(d);
    dim3 dist_grid_dim(ceil(((float) n) / dist_max_points_per_warp), k);
    dim3 dist_block_dim(dist_max_points_per_warp * next_pow_2(d));
    uint32_t dist_kernel_sh_mem = 0;
  #else
    dim3 dist_grid_dim(n, k);
    dim3 dist_block_dim(d);
    uint32_t dist_kernel_sh_mem = 0;
  #endif
  

  /* MAIN LOOP */
  while (iter++ < maxiter) {

    /* COMPUTE DISTANCES */
    CHECK_CUDA_ERROR(hipMemcpy(d_centers, h_centers, CENTERS_BYTES, hipMemcpyHostToDevice));
    if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " compute_distances: Grid (%d, %d, %d), Block (%d, %d, %d), Sh.mem. %uB\n", dist_grid_dim.x, dist_grid_dim.y, dist_grid_dim.z, dist_block_dim.x, dist_block_dim.y, dist_block_dim.z, dist_kernel_sh_mem);
    #if PERFORMANCES_KERNEL_DISTANCES
      hipEvent_t e_perf_dist_start, e_perf_dist_stop;
      hipEventCreate(&e_perf_dist_start);
      hipEventCreate(&e_perf_dist_stop);
      hipEventRecord(e_perf_dist_start);
    #endif
    #if COMPUTE_DISTANCES_KERNEL == 1
      compute_distances_shmem<<<dist_grid_dim, dist_block_dim, dist_kernel_sh_mem>>>(d_distances, d_centers, d_points, dist_max_points_per_warp, d);
    #elif COMPUTE_DISTANCES_KERNEL == 2     
      compute_distances_shfl<<<dist_grid_dim, dist_block_dim>>>(d_distances, d_centers, d_points, n, dist_max_points_per_warp, d, next_pow_2(d));
    #else
      compute_distances_one_point_per_warp<<<dist_grid_dim, dist_block_dim>>>(d_distances, d_centers, d_points);
    #endif
    CHECK_LAST_CUDA_ERROR();
    #if PERFORMANCES_KERNEL_DISTANCES
      hipEventRecord(e_perf_dist_stop);
      hipEventSynchronize(e_perf_dist_stop);
      float e_perf_dist_ms = 0;
      hipEventElapsedTime(&e_perf_dist_ms, e_perf_dist_start, e_perf_dist_stop);
      printf(CYAN "[PERFORMANCE]" RESET " compute_distances time: %.6f\n", e_perf_dist_ms / 1000);
      hipEventDestroy(e_perf_dist_start);
      hipEventDestroy(e_perf_dist_stop);
    #endif

    #if DEBUG_KERNEL_DISTANCES
      printf(GREEN "[DEBUG_KERNEL_DISTANCES]\n");
      DATA_TYPE tmp[n * k];
      CHECK_CUDA_ERROR(hipMemcpy(tmp, d_distances, n * k * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
      hipDeviceSynchronize();
      for (uint32_t i = 0; i < n; ++i)
        for (uint32_t j = 0; j < k; ++j)
          printf("%u %u -> %.3f\n", i, j, tmp[i * k + j]);
      cout << RESET << endl;
    #endif


    /* ASSIGN POINTS TO NEW CLUSTERS */
    #if DEBUG_KERNEL_ARGMIN
      printf("DEBUG_KERNEL_ARGMIN\n");
    #endif
    memset(h_clusters_len, 0, k * sizeof(uint64_t));
    for (size_t i = 0; i < n; i++) {
      hipcub::KeyValuePair<int32_t, DATA_TYPE> *d_argmin = NULL;
      CHECK_CUDA_ERROR(hipMalloc(&d_argmin, sizeof(int32_t) + sizeof(DATA_TYPE)));
      // Allocate temporary storage
      void *d_temp_storage = NULL; size_t temp_storage_bytes = 0;
      hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_distances, d_argmin, k);
      CHECK_CUDA_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));
      
      // Run argmin-reduction
      hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_distances + i * k, d_argmin, k);

      int32_t argmin_idx;
      DATA_TYPE argmin_val;
      CHECK_CUDA_ERROR(hipMemcpy(&argmin_idx, &(d_argmin->key), sizeof(int32_t), hipMemcpyDeviceToHost));
      CHECK_CUDA_ERROR(hipMemcpy(&argmin_val, &(d_argmin->value), sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
      CHECK_CUDA_ERROR(hipFree(d_temp_storage));
      CHECK_CUDA_ERROR(hipFree(d_argmin));
      CHECK_CUDA_ERROR(hipDeviceSynchronize());

      #if DEBUG_KERNEL_ARGMIN
        printf("Argmin point %lu: %d %.3f\n", i, argmin_idx, argmin_val);
      #endif

      ++h_clusters_len[argmin_idx];
      max_cluster_len = max_cluster_len > h_clusters_len[argmin_idx] ? max_cluster_len : h_clusters_len[argmin_idx];
      h_points_clusters[i] = argmin_idx;
    }
    #if DEBUG_KERNEL_ARGMIN
      printf("\n");
    #endif

    CHECK_CUDA_ERROR(hipMemcpy(d_points_clusters, h_points_clusters, n * sizeof(uint32_t), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_clusters_len, h_clusters_len, k * sizeof(uint64_t), hipMemcpyHostToDevice));
    hipMemset(h_centers, 0, k * d * sizeof(DATA_TYPE));

    //printf("POINTS\n");
    for (uint32_t i = 0; i < n; ++i) {
      for (uint32_t j = 0; j < d; ++j) {
        //printf("%.3f, ", h_points[i * d + j]);
        h_centers[h_points_clusters[i] * d + j] += h_points[i * d + j];
      }
      //printf("%d\n", h_points_clusters[i]);
    } 

    #if PERFORMANCES_KERNEL_CENTERS
      hipEvent_t e_perf_cent_start, e_perf_cent_stop;
      hipEventCreate(&e_perf_cent_start);
      hipEventCreate(&e_perf_cent_stop);
      hipEventRecord(e_perf_cent_start);
    #endif

    /* COMPUTE NEW CENTERS */
    // hipMemset(d_centers, 0, k * d * sizeof(DATA_TYPE));
    // dim3 centers_grid_dim(n);
    // if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET "compute_centers: Grid (%d, %d, %d), Block (%d, %d, %d)\n", centers_grid_dim.x, centers_grid_dim.y, centers_grid_dim.z, d, 1, 1);
    // compute_centers<<<centers_grid_dim, d/*, k * d * sizeof(DATA_TYPE) */>>>(d_centers, d_points, d_points_clusters, d_clusters_len);
    // CHECK_LAST_CUDA_ERROR();
    // hipDeviceSynchronize(); 
    
    hipMemset(d_centers, 0, k * d * sizeof(DATA_TYPE));
    dim3 cent_block_dim(n > 32 ? next_pow_2((n + 1) / 2) : 32, d); 
    dim3 cent_grid_dim(k);
    int threads_tot = cent_block_dim.x * cent_block_dim.y;
    while (threads_tot > deviceProps->maxThreadsPerBlock) {
      cent_block_dim.x /= 2;
      cent_grid_dim.y *= 2;
      threads_tot = cent_block_dim.x * cent_block_dim.y;
    }  
    size_t cent_sh_mem = 0;
    if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " compute_centers: Grid (%u, %u, %u), Block (%u, %u, %u), Sh.mem. %luB\n", cent_grid_dim.x, cent_grid_dim.y, cent_grid_dim.z, cent_block_dim.x, cent_block_dim.y, cent_block_dim.z, cent_sh_mem);
    compute_centers_shfl<<<cent_grid_dim, cent_block_dim, cent_sh_mem>>>(d_centers, d_points, d_points_clusters, n, d);
    CHECK_LAST_CUDA_ERROR();
    hipDeviceSynchronize();    

    #if PERFORMANCES_KERNEL_CENTERS
      hipEventRecord(e_perf_cent_stop);
      hipEventSynchronize(e_perf_cent_stop);
      float e_perf_cent_ms = 0;
      hipEventElapsedTime(&e_perf_cent_ms, e_perf_cent_start, e_perf_cent_stop);
      printf(CYAN "[PERFORMANCE]" RESET " compute_centers time: %.6f\n", e_perf_cent_ms / 1000);
      hipEventDestroy(e_perf_cent_start);
      hipEventDestroy(e_perf_cent_stop);
    #endif
    
    CHECK_CUDA_ERROR(hipMemcpy(h_centers, d_centers, d * k * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
    
    for (uint32_t i = 0; i < k; ++i)
      for (uint32_t j = 0; j < d; ++j)
        h_centers[i * d + j] /= h_clusters_len[i];    

    #if DEBUG_KERNEL_CENTERS
      printf("DEBUG_KERNEL_CENTERS\n");
      cout << endl << "CENTERS" << endl;
      for (uint32_t i = 0; i < k; ++i) {
        for (uint32_t j = 0; j < d; ++j)
          printf("%.3f, ", h_centers[i * d + j]);
        cout << endl;
      }
      cout << endl;
    #endif

    /* CHECK IF CONVERGED */
    if (iter > 1 && cmpCenters()) { // Exit
      converged = iter;
      break;
    } else { // Copy centers
      memcpy(h_last_centers, h_centers, CENTERS_BYTES);
    }
  }
  /* MAIN LOOP END */

  /* COPY BACK RESULTS*/
  for (size_t i = 0; i < n; i++) {
    points[i]->setCluster(h_points_clusters[i]);
  }
  

  /* FREE MEMORY */
  CHECK_CUDA_ERROR(hipMalloc(&d_clusters_len, k * sizeof(uint64_t)));
  CHECK_CUDA_ERROR(hipFree(d_distances));
  CHECK_CUDA_ERROR(hipFree(d_points_clusters));
  CHECK_CUDA_ERROR(hipFree(d_clusters_len));
  CHECK_CUDA_ERROR(hipHostFree(h_clusters_len));

  return converged;
}

bool Kmeans::cmpCenters () {
  const DATA_TYPE EPSILON = numeric_limits<DATA_TYPE>::epsilon();
  DATA_TYPE dist_sum = 0, norm = 0;
  
  for (size_t i = 0; i < k; ++i) {
    for (size_t j = 0; j < d; ++j) {
      DATA_TYPE dist = fabs(h_centers[i * d + j] - h_last_centers[i * d + j]);
      dist_sum += dist * dist;
      norm += h_last_centers[i * d + j] * h_last_centers[i * d + j];
    }
    if (sqrt(dist_sum) > EPSILON) { return false; }
  }

  return true;
}

void Kmeans::to_csv(ostream& o, char separator) {
  o << "cluster" << separator;
  for (size_t i = 0; i < d; ++i) {
    o << "d" << i;
    if (i != (d - 1)) o << separator;
  }
  o << endl;
  for (size_t i = 0; i < n; ++i) {
    o << h_points_clusters[i] << separator;
    for (size_t j = 0; j < d; ++j) {
      o << setprecision(8) << h_points[i * d + j];
      if (j != (d - 1)) o << separator;
    }
    o << endl;
  }
}