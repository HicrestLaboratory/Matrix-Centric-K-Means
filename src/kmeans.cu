#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <random>
#include <iomanip>
#include <cmath>
#include <limits>
#include <hipblas.h>

#include "include/common.h"
#include "include/colors.h"

#include "utils.cuh"
#include "kmeans.cuh"

#include "kernels/kernels.cuh"

using namespace std;

random_device rd;
seed_seq seed{0}; // FIXME use rd()
mt19937 rng(seed);

const DATA_TYPE INFNTY  = numeric_limits<DATA_TYPE>::infinity();

/* Kmeans class */
void Kmeans::init_centroids (Point<DATA_TYPE>** points) {
  uniform_int_distribution<int> random_int(0, n - 1);
  if (COMPUTE_DISTANCES_KERNEL >= 2) {
    CENTROIDS_BYTES += (k * sizeof(DATA_TYPE)); // Be aware
    CHECK_CUDA_ERROR(hipHostAlloc(&h_centroids_matrix, CENTROIDS_BYTES, hipHostMallocDefault));
  } else {
    h_centroids_matrix = NULL;
  }
  CHECK_CUDA_ERROR(hipHostAlloc(&h_centroids, CENTROIDS_BYTES, hipHostMallocDefault));
  CHECK_CUDA_ERROR(hipHostAlloc(&h_last_centroids, CENTROIDS_BYTES, hipHostMallocDefault));
  unsigned int i = 0;
  vector<Point<DATA_TYPE>*> usedPoints;
  Point<DATA_TYPE>* centroids[k];
  while (i < k) {
    Point<DATA_TYPE>* p = points[random_int(rng)];
    bool found = false;
    for (auto p1 : usedPoints) {
      if ((*p1) == (*p)) { // FIXME Is it better use some min distance??
        found = true;
        break;
      }
    }
    if (!found) {
      centroids[i] = new Point<DATA_TYPE>(p);
      usedPoints.push_back(p);
      ++i;
    }
  }

  #if DEBUG_INIT_CENTROIDS
    cout << endl << "Centroids" << endl;
    for (i = 0; i < k; ++i)
      cout << *(centroids[i]) << endl;
  #endif

  for (size_t i = 0; i < k; ++i) {
    for (size_t j = 0; j < d; ++j) {
      h_centroids[i * d + j] = centroids[i]->get(j);                // Row major
      #if COMPUTE_DISTANCES_KERNEL >= 2
        h_centroids_matrix[(j + 1) * k + i] = centroids[i]->get(j); // Col major
      #endif
    }
  }
  #if COMPUTE_DISTANCES_KERNEL >= 2
    for (size_t i = 0; i < k; ++i)
      h_centroids_matrix[i] = 1; // Static prefix
  #endif
  CHECK_CUDA_ERROR(hipMalloc(&d_centroids, CENTROIDS_BYTES));
}

Kmeans::Kmeans (size_t _n, unsigned int _d, unsigned int _k, float _tol, Point<DATA_TYPE>** _points, hipDeviceProp_t* _deviceProps)
    : n(_n), d(_d), k(_k), tol(_tol),
    POINTS_BYTES(_n * _d * sizeof(DATA_TYPE)),
    CENTROIDS_BYTES(_k * _d * sizeof(DATA_TYPE)),
    points(_points),
    deviceProps(_deviceProps) {

  CHECK_CUDA_ERROR(hipHostAlloc(&h_points, POINTS_BYTES, hipHostMallocDefault));
  for (size_t i = 0; i < n; ++i) {
    for (size_t j = 0; j < d; ++j) {
      h_points[i * d + j] = _points[i]->get(j);
    }
  }
  CHECK_CUDA_ERROR(hipMalloc(&d_points, POINTS_BYTES));
  CHECK_CUDA_ERROR(hipMemcpy(d_points, h_points, POINTS_BYTES, hipMemcpyHostToDevice));

  init_centroids(_points);
  CHECK_CUDA_ERROR(hipMemcpy(d_centroids, h_centroids, d * k * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
}

Kmeans::~Kmeans () {
  CHECK_CUDA_ERROR(hipHostFree(h_points));
  CHECK_CUDA_ERROR(hipHostFree(h_centroids));
  CHECK_CUDA_ERROR(hipHostFree(h_last_centroids));
  CHECK_CUDA_ERROR(hipHostFree(h_points_clusters));
  CHECK_CUDA_ERROR(hipFree(d_centroids));
  CHECK_CUDA_ERROR(hipFree(d_points));
  if (h_centroids_matrix != NULL) {
    CHECK_CUDA_ERROR(hipHostFree(h_centroids_matrix));
  }
  compute_gemm_distances_free();
}

uint64_t Kmeans::run (uint64_t maxiter) {
  uint64_t converged = maxiter;

  /* INIT */
  DATA_TYPE* d_distances;
  CHECK_CUDA_ERROR(hipMalloc(&d_distances, n * k * sizeof(DATA_TYPE)));
  uint32_t* d_points_clusters;
  CHECK_CUDA_ERROR(hipMalloc(&d_points_clusters, n * sizeof(uint32_t)));
  CHECK_CUDA_ERROR(hipHostMalloc(&h_points_clusters, n * sizeof(uint32_t)));
  uint32_t* d_clusters_len;
  CHECK_CUDA_ERROR(hipMalloc(&d_clusters_len, k * sizeof(uint32_t)));

  uint64_t iter = 0;
  const uint32_t rounds = ((d - 1) / deviceProps->warpSize) + 1;

  #if COMPUTE_DISTANCES_KERNEL <= 1
    dim3 dist_grid_dim, dist_block_dim;
    uint32_t dist_max_points_per_warp;
    schedule_distances_kernel(deviceProps, n, d, k, &dist_grid_dim, &dist_block_dim, &dist_max_points_per_warp);
  #else
    DATA_TYPE* d_points_assoc_matrices;
    DATA_TYPE* d_centroids_matrix;
    uint32_t d1 = d + 1;
    uint32_t nd1d1 = n * d1 * d1;
    // Associated to POINTS (centers change after every iteration)
    CHECK_CUDA_ERROR(hipMalloc(&d_points_assoc_matrices, nd1d1 * sizeof(DATA_TYPE)));
    CHECK_CUDA_ERROR(hipMemset(d_points_assoc_matrices, 0, nd1d1 * sizeof(DATA_TYPE)));
    CHECK_CUDA_ERROR(hipMalloc(&d_centroids_matrix, CENTROIDS_BYTES));
    dim3 dist_assoc_matrices_grid_dim(n);
    dim3 dist_assoc_matrices_block_dim(min(next_pow_2(d), deviceProps->warpSize));
    #if COMPUTE_DISTANCES_KERNEL >= 2 && DEBUG_KERNELS_INVOKATION
      printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u), Sh.mem. %uB\n", "compute_point_associated_matrices", dist_assoc_matrices_grid_dim.x, dist_assoc_matrices_grid_dim.y, dist_assoc_matrices_grid_dim.z, dist_assoc_matrices_block_dim.x, dist_assoc_matrices_block_dim.y, dist_assoc_matrices_block_dim.z, 0);
    #endif
    for (uint32_t i = 0; i < rounds; i++) {
      compute_point_associated_matrices<<<dist_assoc_matrices_grid_dim, dist_assoc_matrices_block_dim>>>(d_points, d_points_assoc_matrices, d, i);
    }
    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS_ERROR(hipblasCreate(&cublasHandle));
  #endif

  dim3 argmin_grid_dim, argmin_block_dim;
  uint32_t argmin_warps_per_block, argmin_sh_mem;
  schedule_argmin_kernel(deviceProps, n, k, &argmin_grid_dim, &argmin_block_dim, &argmin_warps_per_block, &argmin_sh_mem);

  dim3 cent_grid_dim, cent_block_dim;
  schedule_centroids_kernel(deviceProps, n, d, k, &cent_grid_dim, &cent_block_dim);

  /* MAIN LOOP */
  while (iter++ < maxiter) {
    /* COMPUTE DISTANCES */

    #if COMPUTE_DISTANCES_KERNEL >= 2
      if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " Matmul\n");
    #else
      if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u), Sh.mem. %uB\n", "compute_distances", dist_grid_dim.x, dist_grid_dim.y, dist_grid_dim.z, dist_block_dim.x, dist_block_dim.y, dist_block_dim.z, 0);
    #endif
    #if PERFORMANCES_KERNEL_DISTANCES
      hipEvent_t e_perf_dist_start, e_perf_dist_stop;
      hipEventCreate(&e_perf_dist_start);
      hipEventCreate(&e_perf_dist_stop);
      hipEventRecord(e_perf_dist_start);
    #endif
    #if COMPUTE_DISTANCES_KERNEL <= 1
      if (static_cast<int>(d) <= deviceProps->warpSize && COMPUTE_DISTANCES_KERNEL == 1) {
        compute_distances_shfl<<<dist_grid_dim, dist_block_dim>>>(d_distances, d_centroids, d_points, n, dist_max_points_per_warp, d, log2(next_pow_2(d)) > 0 ? log2(next_pow_2(d)) : 1);
      } else {
        for (uint32_t i = 0; i < rounds; i++) {
          compute_distances_one_point_per_warp<<<dist_grid_dim, dist_block_dim>>>(d_distances, d_centroids, d_points, d, next_pow_2(d), i);
        }
      }
    #else
      CHECK_CUBLAS_ERROR(hipblasSetMatrix(k, d1, sizeof(DATA_TYPE), h_centroids_matrix, k, d_centroids_matrix, k)); // same as CHECK_CUDA_ERROR(hipMemcpy(d_centroids_matrix, h_centroids_matrix, CENTROIDS_BYTES, hipMemcpyHostToDevice));
      compute_gemm_distances(cublasHandle, d1, n, k, d_points_assoc_matrices, d_centroids_matrix, d_distances);
    #endif
    #if PERFORMANCES_KERNEL_DISTANCES
      hipEventRecord(e_perf_dist_stop);
      hipEventSynchronize(e_perf_dist_stop);
      float e_perf_dist_ms = 0;
      hipEventElapsedTime(&e_perf_dist_ms, e_perf_dist_start, e_perf_dist_stop);
      printf(CYAN "[PERFORMANCE]" RESET " compute_distances time: %.8f\n", e_perf_dist_ms / 1000);
      hipEventDestroy(e_perf_dist_start);
      hipEventDestroy(e_perf_dist_stop);
    #endif

    #if DEBUG_KERNEL_DISTANCES
      printf(GREEN "[DEBUG_KERNEL_DISTANCES]\n");
      #if COMPUTE_DISTANCES_KERNEL >= 2
        cout << "Centroids matrix" << endl;
        printMatrixColMaj(h_centroids_matrix, k, d1);
        cout << endl;
        DATA_TYPE tmp_assoc_mat[(d + 1) * (d + 1)];
        uint32_t d1d1 = d1 * d1;
        for (size_t i = 0; i < 1; i++) {
          cout << "Point " << i << " associated matrix" << endl;
          CHECK_CUDA_ERROR(hipMemcpy(tmp_assoc_mat, d_points_assoc_matrices + (d1d1 * i), d1d1 * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
          printMatrixColMaj(tmp_assoc_mat, d1, d1);
          cout << endl;
        }
      #endif
      DATA_TYPE* cpu_distances = new DATA_TYPE[n * k];
      for (uint32_t ni = 0; ni < n; ++ni) {
        for (uint32_t ki = 0; ki < k; ++ki) {
          DATA_TYPE dist = 0, tmp;
          for (uint32_t di = 0; di < d; ++di) {
            tmp = h_points[ni * d + di] - h_centroids[ki * d + di];
            dist += tmp * tmp;
          }
          cpu_distances[ni * k + ki] = dist;
        }
      }
      DATA_TYPE* tmp_dist = new DATA_TYPE[n * k];
      int anyError = 0;
      CHECK_CUDA_ERROR(hipMemcpy(tmp_dist, d_distances, n * k * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
      for (uint32_t i = 0; i < n; ++i)
        for (uint32_t j = 0; j < k; ++j)
          if (fabs(tmp_dist[i * k + j] - cpu_distances[i * k + j]) > 0.001) {
            printf("N=%-2u K=%-2u -> GPU=%.4f CPU=%.4f diff: %.8f\n", i, j, tmp_dist[i * k + j], cpu_distances[i * k + j], fabs(tmp_dist[i * k + j] - cpu_distances[i * k + j]));
            anyError = 1;
          }
      cout << (anyError ? "Something wrong" : "Everything alright") << RESET << endl;
      delete[] cpu_distances;
      delete[] tmp_dist;
    #endif

    ////////////////////////////////////////* ASSIGN POINTS TO NEW CLUSTERS */////////////////////////////////////////

    #if PERFORMANCES_KERNEL_ARGMIN
      hipEvent_t e_perf_argmin_start, e_perf_argmin_stop;
      hipEventCreate(&e_perf_argmin_start);
      hipEventCreate(&e_perf_argmin_stop);
      hipEventRecord(e_perf_argmin_start);
    #endif

    if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u), Sh.mem. %uB\n", "clusters_argmin_shfl", argmin_grid_dim.x, argmin_grid_dim.y, argmin_grid_dim.z, argmin_block_dim.x, argmin_block_dim.y, argmin_block_dim.z, argmin_sh_mem);

    CHECK_CUDA_ERROR(hipMemset(d_clusters_len, 0, k * sizeof(uint32_t)));
    clusters_argmin_shfl<<<argmin_grid_dim, argmin_block_dim, argmin_sh_mem>>>(n, k, d_distances, d_points_clusters, d_clusters_len, argmin_warps_per_block, INFNTY);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    #if PERFORMANCES_KERNEL_ARGMIN
      hipEventRecord(e_perf_argmin_stop);
      hipEventSynchronize(e_perf_argmin_stop);
      float e_perf_argmin_ms = 0;
      hipEventElapsedTime(&e_perf_argmin_ms, e_perf_argmin_start, e_perf_argmin_stop);
      printf(CYAN "[PERFORMANCE]" RESET " clusters_argmin_shfl time: %.8f\n", e_perf_argmin_ms / 1000);
      hipEventDestroy(e_perf_argmin_stop);
      hipEventDestroy(e_perf_argmin_start);
    #endif

    #if DEBUG_KERNEL_ARGMIN
      printf(GREEN "[DEBUG_KERNEL_ARGMIN]\n" RESET);
      uint32_t tmp1[n];
      CHECK_CUDA_ERROR(hipMemcpy(tmp1, d_points_clusters, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
      printf(GREEN "p  -> c\n");
      for (uint32_t i = 0; i < n; ++i)
          printf("%-2u -> %-2u\n", i, tmp1[i]);
      cout << RESET << endl;
    #endif

    ///////////////////////////////////////////* COMPUTE NEW CENTROIDS *///////////////////////////////////////////

    CHECK_CUDA_ERROR(hipMemset(h_centroids, 0, k * d * sizeof(DATA_TYPE)));
    CHECK_CUDA_ERROR(hipMemset(d_centroids, 0, k * d * sizeof(DATA_TYPE)));

    #if PERFORMANCES_KERNEL_CENTROIDS
      hipEvent_t e_perf_cent_start, e_perf_cent_stop;
      hipEventCreate(&e_perf_cent_start);
      hipEventCreate(&e_perf_cent_stop);
      hipEventRecord(e_perf_cent_start);
    #endif

    if (DEBUG_KERNELS_INVOKATION) printf(YELLOW "[KERNEL]" RESET " %-25s: Grid (%4u, %4u, %4u), Block (%4u, %4u, %4u)\n", "compute_centroids", cent_grid_dim.x, cent_grid_dim.y, cent_grid_dim.z, cent_block_dim.x, cent_block_dim.y, cent_block_dim.z);

    for (uint32_t i = 0; i < rounds; i++) {
      compute_centroids_shfl<<<cent_grid_dim, cent_block_dim>>>(d_centroids, d_points, d_points_clusters, d_clusters_len, n, d, k, i);
    }
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    #if PERFORMANCES_KERNEL_CENTROIDS
      hipEventRecord(e_perf_cent_stop);
      hipEventSynchronize(e_perf_cent_stop);
      float e_perf_cent_ms = 0;
      hipEventElapsedTime(&e_perf_cent_ms, e_perf_cent_start, e_perf_cent_stop);
      printf(CYAN "[PERFORMANCE]" RESET " compute_centroids time: %.8f\n", e_perf_cent_ms / 1000);
      hipEventDestroy(e_perf_cent_start);
      hipEventDestroy(e_perf_cent_stop);
    #endif

    #if DEBUG_KERNEL_CENTROIDS
      uint32_t* h_clusters_len;
      CHECK_CUDA_ERROR(hipHostMalloc(&h_clusters_len, k * sizeof(uint32_t)));
      CHECK_CUDA_ERROR(hipMemcpy(h_points_clusters, d_points_clusters, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
      CHECK_CUDA_ERROR(hipMemcpy(h_clusters_len,    d_clusters_len,    k * sizeof(uint32_t), hipMemcpyDeviceToHost));
      for (uint32_t i = 0; i < n; ++i) {
        for (uint32_t j = 0; j < d; ++j) {
          h_centroids[h_points_clusters[i] * d + j] += h_points[i * d + j];
        }
      }
      for (uint32_t i = 0; i < k; ++i) {
        for (uint32_t j = 0; j < d; ++j) {
          uint64_t count = h_clusters_len[i] > 1 ? h_clusters_len[i] : 1;
          DATA_TYPE scale = 1.0 / ((double) count);
          h_centroids[i * d + j] *= scale;
        }
      }
      cout << GREEN "[DEBUG_KERNEL_CENTROIDS]" << endl;
      cout << endl << "CENTROIDS (CPU)" << endl;
      for (uint32_t i = 0; i < k; ++i) {
        for (uint32_t j = 0; j < d; ++j)
          printf("%.3f, ", h_centroids[i * d + j]);
        cout << endl;
      }
      CHECK_CUDA_ERROR(hipMemset(h_centroids, 0, d * k * sizeof(DATA_TYPE)));
      CHECK_CUDA_ERROR(hipMemcpy(h_centroids, d_centroids, d * k * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
      cout << endl << "CENTROIDS (GPU)" << endl;
      for (uint32_t i = 0; i < k; ++i) {
        for (uint32_t j = 0; j < d; ++j)
          printf("%.3f, ", h_centroids[i * d + j]);
        cout << endl;
      }
      cout << RESET << endl;
      CHECK_CUDA_ERROR(hipHostFree(h_clusters_len));
    #endif

    CHECK_CUDA_ERROR(hipMemcpy(h_centroids, d_centroids, d * k * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));

    /////////////////////////////////////////////* CHECK IF CONVERGED */////////////////////////////////////////////

    // Check exit
    if (iter > 1 && cmp_centroids()) {
      converged = iter;
      break;
    }

    // Copy current centroids
    memcpy(h_last_centroids, h_centroids, CENTROIDS_BYTES);

    #if COMPUTE_DISTANCES_KERNEL >= 2
      /* UPDATE h_centroids_matrix */
      for (size_t i = 0; i < k; ++i) {
        h_centroids_matrix[i] = 1; // Static prefix
        for (size_t j = 0; j < d; ++j) {
          h_centroids_matrix[IDX2C(i, j + 1, k)] = h_centroids[i * d + j]; // Row maj to Col maj
        }
      }
    #endif

  }
  /* MAIN LOOP END */

  /* COPY BACK RESULTS*/
  CHECK_CUDA_ERROR(hipMemcpy(h_points_clusters, d_points_clusters, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
  for (size_t i = 0; i < n; i++) {
    points[i]->setCluster(h_points_clusters[i]);
  }

  /* FREE MEMORY */
  CHECK_CUDA_ERROR(hipFree(d_distances));
  CHECK_CUDA_ERROR(hipFree(d_points_clusters));
  CHECK_CUDA_ERROR(hipFree(d_clusters_len));
  #if COMPUTE_DISTANCES_KERNEL >= 2
    CHECK_CUDA_ERROR(hipFree(d_points_assoc_matrices));
    CHECK_CUDA_ERROR(hipFree(d_centroids_matrix));
    CHECK_CUBLAS_ERROR(hipblasDestroy(cublasHandle));
  #endif

  return converged;
}

bool Kmeans::cmp_centroids () {
  for (size_t i = 0; i < k; ++i) {
    DATA_TYPE dist_sum = 0;
    for (size_t j = 0; j < d; ++j) {
      DATA_TYPE dist = h_centroids[i * d + j] - h_last_centroids[i * d + j];
      dist_sum += dist * dist;
    }
    if (sqrt(dist_sum) > tol) { return false; }
  }

  return true;
}

void Kmeans::to_csv(ostream& o, char separator) {
  o << "cluster" << separator;
  for (size_t i = 0; i < d; ++i) {
    o << "d" << i;
    if (i != (d - 1)) o << separator;
  }
  o << endl;
  for (size_t i = 0; i < n; ++i) {
    o << h_points_clusters[i] << separator;
    for (size_t j = 0; j < d; ++j) {
      o << setprecision(8) << h_points[i * d + j];
      if (j != (d - 1)) o << separator;
    }
    o << endl;
  }
}