#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

#include "kernels.cuh"
#include "../utils.cuh"

#define DEBUG_GEMM 0

/*** Warp oriented ***/

__global__ void compute_distances_one_point_per_warp(DATA_TYPE* distances, DATA_TYPE* centroids, DATA_TYPE* points, uint32_t next_pow_2) {
  const uint64_t point_offset = blockIdx.x * blockDim.x + threadIdx.x;
  const uint64_t center_offset = blockIdx.y * blockDim.x + threadIdx.x;
  DATA_TYPE dist = points[point_offset] - centroids[center_offset];
  dist *= dist;
  
  for (int i = next_pow_2; i > 0; i /= 2)
    dist += __shfl_down_sync(DISTANCES_SHFL_MASK, dist, i);

  if (threadIdx.x == 0) {
    distances[(blockIdx.x * gridDim.y) + blockIdx.y] = dist;
  }
}

__global__ void compute_distances_shmem(DATA_TYPE* distances, DATA_TYPE* centroids, DATA_TYPE* points, const uint32_t points_per_warp, const uint32_t d) {
  const uint64_t point_i = (blockIdx.x * points_per_warp) + (threadIdx.x / d);
  const uint64_t center_i = blockIdx.y;
  const uint32_t d_i = threadIdx.x % d;
  const uint64_t dists_i = (center_i * blockDim.y * d) + ((point_i % points_per_warp) * d) + d_i;

  extern __shared__ DATA_TYPE dists[];

  if (threadIdx.x < points_per_warp * d) {
    DATA_TYPE dist = fabs(points[point_i * d + d_i] - centroids[center_i * d + d_i]);
    dists[dists_i] = dist * dist;
    __syncthreads();
    if (d_i == 0) {
      for (int i = 1; i < d; i++) {
        dists[dists_i] += dists[dists_i + i];
      }
      distances[(point_i * center_i) + point_i] = dists[dists_i];
    }
  }
}

__global__ void compute_distances_shfl(DATA_TYPE* distances, DATA_TYPE* centroids, DATA_TYPE* points, const uint32_t points_n, const uint32_t points_per_warp, const uint32_t d, const uint32_t d_closest_2_pow) {
  const uint32_t point_i = (blockIdx.x * points_per_warp) + (threadIdx.x / d_closest_2_pow);
  const uint32_t center_i = blockIdx.y;
  const uint32_t d_i = threadIdx.x % d_closest_2_pow;

  if (point_i < points_n && d_i < d) {
    DATA_TYPE dist = fabs(points[point_i * d + d_i] - centroids[center_i * d + d_i]);
    dist *= dist;
    for (int i = d_closest_2_pow / 2; i > 0; i /= 2) {
      dist += __shfl_down_sync(DISTANCES_SHFL_MASK, dist, i);
      // if (point_i == 3) printf("%d  p: %lu c: %lu d: %u v: %.3f\n", i, point_i, center_i, d_i, dist);
    }
    if (d_i == 0) {
      distances[(point_i * gridDim.y) + center_i] = dist;
    }
  }
}

/*** END Warp oriented ***/

/*** Matrix multiplication ***/
/**
 * NOTICE: the reduction limits the maximum block size to 32 (warpSize) 
*/
__global__ void compute_point_associated_matrices (const DATA_TYPE* points, DATA_TYPE* associated_matrices, const uint32_t d, const uint32_t round) {
  const uint32_t block_base = warpSize * round;
  const uint32_t p_i = blockIdx.x;
  const uint32_t d_i = block_base + threadIdx.x;
  const uint32_t d_i1 = d_i + 1;

  // If dim in the thread is greater than d, then return to avoid illegal writes
  if (d_i >= d) { return; } 

  DATA_TYPE c = points[p_i * d + d_i];
  DATA_TYPE c_11 = c * c;

  for (int i = warpSize / 2; i > 0; i /= 2) { // Reduce c_11
    c_11 += __shfl_down_sync(DISTANCES_SHFL_MASK, c_11, i);
  }

  const uint32_t d1 = d + 1;
  const uint32_t matrix_base_i = p_i * d1 * d1;
  if (threadIdx.x == 0) {
    atomicAdd(&associated_matrices[matrix_base_i], c_11); // Write reduced c_11
  }
  associated_matrices[matrix_base_i + d_i1] = -c;               // Write first column
  associated_matrices[matrix_base_i + (d_i1 * d1)] = -c;        // Write first row
  associated_matrices[matrix_base_i + (d_i1 * d1) + d_i1] = 1;  // Write diagonal
}

DATA_TYPE* d_tmp = NULL; // https://docs.nvidia.com/cuda/cublas/index.html#cublas-t-gemm
/**
 * @brief Computes and writes to d_distances TODO
 * 
 * @param handle 
 * @param d1 
 * @param n 
 * @param k 
 * @param d_P the points associated matrices
 * @param d_C the matrix of centers (prefixed with 1s)
 * @param d_distances 
 */
void compute_gemm_distances (hipblasHandle_t& handle, uint32_t d1, uint32_t n, uint32_t k, DATA_TYPE* d_P, DATA_TYPE* d_C, DATA_TYPE* d_distances) {
  DATA_TYPE alpha = (DATA_TYPE)1;
  DATA_TYPE beta = (DATA_TYPE)0;
  uint32_t d1d1 = d1 * d1;
  DATA_TYPE* P = d_P;
  uint32_t max_k_d1 = max(k, d1);
  DATA_TYPE h_distances[k * n];
  DATA_TYPE h_tmp[max_k_d1 * max_k_d1];
  if (d_tmp == NULL) {
    hipMalloc(&d_tmp, max_k_d1 * max_k_d1 * sizeof(DATA_TYPE));
  }

  for (uint32_t p_i = 0; p_i < n; ++p_i, P += d1d1) { // Iterate over points associated matrices
    #if DEBUG_GEMM
      printf("\nc\n");
      DATA_TYPE tmp_debug1[n * d1];
      CHECK_CUBLAS_ERROR(hipblasGetMatrix(k, d1, sizeof(DATA_TYPE), d_C, k, tmp_debug1, k));
      printMatrixColMaj(tmp_debug1, k, d1);
      printf("\nP_%d associated matrix\n", p_i);
      DATA_TYPE tmp_debug[d1d1];
      CHECK_CUBLAS_ERROR(hipblasGetMatrix(d1, d1, sizeof(DATA_TYPE), P, d1, tmp_debug, d1));
      printMatrixColMaj(tmp_debug, d1, d1);
      printf("\n");
    #endif
    
    CHECK_CUBLAS_ERROR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, // c * P
                                    k, d1, d1, &alpha,
                                    d_C, k,
                                    P, d1,
                                    &beta, d_tmp, k));

    #if DEBUG_GEMM
      printf("\nc * P\n");
      DATA_TYPE tmp_debug2[k * d1];
      CHECK_CUBLAS_ERROR(hipblasGetMatrix(k, d1, sizeof(DATA_TYPE), d_tmp, k, tmp_debug2, k));
      printMatrixColMaj(tmp_debug2, k, d1);
      printf("\n");
    #endif

    CHECK_CUBLAS_ERROR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, // (c * P) * c^T
                                    k, k, d1, &alpha,
                                    d_tmp, k,
                                    d_C, k,
                                    &beta, d_tmp, k));
    
    
    for (size_t i = 0; i < k; i++) {
      CHECK_CUBLAS_ERROR(hipblasGetMatrix(k, k, sizeof(DATA_TYPE), d_tmp, k, h_tmp, k));
      h_distances[p_i * k + i] = h_tmp[IDX2C(i, i, k)];
    }

    #if DEBUG_GEMM
      printf("Distances from P_%d\n", p_i);
      printMatrixColMaj(h_tmp, k, k);
      printf("\n----------\n");
    #endif
  }
  // Copy distances to GPU
  CHECK_CUDA_ERROR(hipMemcpy(d_distances, h_distances, n * k * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
}

/*** END Matrix multiplication ***/