#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "kernels.cuh"
#include "../utils.cuh"
#include "../include/colors.h"

__device__ Pair shfl_xor_sync (Pair p, unsigned delta){
  return Pair{
    __shfl_xor_sync(ARGMIN_SHFL_MASK, p.v, delta),
    __shfl_xor_sync(ARGMIN_SHFL_MASK, p.i, delta),
  };
}

__device__ Pair argmin (Pair a, Pair b) {
  return a.v <= b.v ? a : b;
}

__device__ Pair warp_argmin (float a) {
  Pair t{a, (uint32_t)threadIdx.x & 31};

  t = argmin(t, shfl_xor_sync(t, 1));
  t = argmin(t, shfl_xor_sync(t, 2));
  t = argmin(t, shfl_xor_sync(t, 4));
  t = argmin(t, shfl_xor_sync(t, 8));
  t = argmin(t, shfl_xor_sync(t, 16));
  return t;
}

__global__ void clusters_argmin_shfl(const uint32_t n, const uint32_t k, DATA_TYPE* d_distances, uint32_t* points_clusters,  uint32_t* clusters_len, uint32_t warps_per_block, DATA_TYPE infty) {
  extern __shared__ Pair shrd[];
  const uint32_t tid = threadIdx.x;
  const uint32_t lane = tid % warpSize;
  const uint32_t wid = tid / warpSize;
  const uint32_t idx = blockIdx.x * k + tid;
  float val = tid < k ? d_distances[idx] : infty;

  Pair p = warp_argmin(val);

  if (lane == 0) {
    p.i += 32 * wid; // Remap p.i
    shrd[wid] = p;
  }
  
  __syncthreads();


  if (tid == 0) { // Intra-block reduction
    Pair* tmp = shrd;
    float minV = tmp->v;
    uint32_t minI = tmp->i;
    for (uint32_t i = 1; i < warps_per_block; i++) {
      Pair* tmp = shrd + i;
      if (tmp->v < minV) {
        minV = tmp->v;
        minI = tmp->i;
      }
    }
    points_clusters[blockIdx.x] = minI;
    atomicAdd(&clusters_len[minI], 1);
  }
}

void clusters_argmin_cub(const DATA_TYPE* d_distances, const uint32_t n, const uint32_t k, uint32_t* h_points_clusters, uint32_t* d_points_clusters, uint64_t* h_clusters_len) {
  memset(h_clusters_len, 0, k * sizeof(uint64_t));
  for (size_t i = 0; i < n; i++) {
    hipcub::KeyValuePair<int32_t, DATA_TYPE> *d_argmin = NULL;
    CHECK_CUDA_ERROR(hipMalloc(&d_argmin, sizeof(int32_t) + sizeof(DATA_TYPE)));
    // Allocate temporary storage
    void *d_temp_storage = NULL; size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_distances, d_argmin, k);
    CHECK_CUDA_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));
    
    // Run argmin-reduction
    hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_distances + i * k, d_argmin, k);

    int32_t argmin_idx;
    DATA_TYPE argmin_val;
    CHECK_CUDA_ERROR(hipMemcpy(&argmin_idx, &(d_argmin->key), sizeof(int32_t), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&argmin_val, &(d_argmin->value), sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipFree(d_temp_storage));
    CHECK_CUDA_ERROR(hipFree(d_argmin));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    ++h_clusters_len[argmin_idx];
    h_points_clusters[i] = argmin_idx;
  }
  CHECK_CUDA_ERROR(hipMemcpy(d_points_clusters, h_points_clusters, n * sizeof(uint32_t), hipMemcpyHostToDevice));
}

void schedule_argmin_kernel(const hipDeviceProp_t *props, const uint32_t n, const uint32_t k, dim3 *grid, dim3 *block, uint32_t *warps_per_block, uint32_t *sh_mem) {
  dim3 argmin_grid_dim(n);
  dim3 argmin_block_dim(max(next_pow_2(k), props->warpSize));
  
  *grid   = argmin_grid_dim;
  *block  = argmin_block_dim;
  *warps_per_block = (k + props->warpSize - 1) / props->warpSize; // Ceil
  *sh_mem = (*warps_per_block) * sizeof(Pair);
}