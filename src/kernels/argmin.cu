#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hipcub/hipcub.hpp>
#include "kernels.cuh"
#include "../utils.cuh"
#include "../include/colors.h"

__device__ Pair shfl_xor_sync (Pair p, unsigned delta){
  return Pair{
    __shfl_xor_sync(ARGMIN_SHFL_MASK, p.v, delta),
    __shfl_xor_sync(ARGMIN_SHFL_MASK, p.i, delta),
  };
}

__device__ Pair argmin (Pair a, Pair b) {
  return a.v <= b.v ? a : b;
}

__device__ Pair warp_argmin (float a) {
  Pair t{a, (uint32_t)threadIdx.x & 31};
  t = argmin(t, shfl_xor_sync(t, 1));
  t = argmin(t, shfl_xor_sync(t, 2));
  t = argmin(t, shfl_xor_sync(t, 4));
  t = argmin(t, shfl_xor_sync(t, 8));
  t = argmin(t, shfl_xor_sync(t, 16));
  return t;
}

/**
 * @brief This kernel reduces each block (one per point) to find the closest centroid (min dist.) and writes back the centroid index incrementing the cluster length
 * 
 * @param n 
 * @param k 
 * @param d_distances 
 * @param points_clusters point-cluster associations
 * @param clusters_len length of clusters
 * @param warps_per_block used to avoid useless compoutations
 * @param infty max value for DATA_TYPE
 */
__global__ void clusters_argmin_shfl(const uint32_t n, const uint32_t k, DATA_TYPE* d_distances, uint32_t* points_clusters,  uint32_t* clusters_len, uint32_t warps_per_block, DATA_TYPE infty) {
  const uint32_t warpSizeLog2 = sizeof(uint32_t) * CHAR_BIT - clz(warpSize) - 1;
  extern __shared__ Pair shrd[];
  const uint32_t tid = threadIdx.x;
  const uint32_t lane = tid & (warpSize - 1);
  const uint32_t wid = tid >> warpSizeLog2;
  const uint32_t idx = blockIdx.x * k + tid;
  float val = tid < k ? d_distances[idx] : infty;

  Pair p = warp_argmin(val);

  if (lane == 0) {
    p.i += wid << warpSizeLog2; // Remap p.i
    shrd[wid] = p;
  }
  
  __syncthreads();


  if (tid == 0) { // Intra-block reduction
    Pair* tmp = shrd;
    float minV = tmp->v;
    uint32_t minI = tmp->i;
    for (uint32_t i = 1; i < warps_per_block; i++) {
      Pair* tmp = shrd + i;
      if (tmp->v < minV) {
        minV = tmp->v;
        minI = tmp->i;
      }
    }
    points_clusters[blockIdx.x] = minI;
    atomicAdd(&clusters_len[minI], 1);
  }
}

/**
 * @brief This function uses the library CUB to perform the argmin for each point/centers
 * 
 * @param d_distances 
 * @param n 
 * @param k 
 * @param h_points_clusters indicates the cluster of each point
 * @param d_points_clusters 
 * @param h_clusters_len indicates how many point belog to each cluster
 */
void clusters_argmin_cub(const DATA_TYPE* d_distances, const uint32_t n, const uint32_t k, uint32_t* h_points_clusters, uint32_t* d_points_clusters, uint64_t* h_clusters_len) {
  memset(h_clusters_len, 0, k * sizeof(uint64_t));
  for (size_t i = 0; i < n; i++) {
    hipcub::KeyValuePair<int32_t, DATA_TYPE> *d_argmin = NULL;
    CHECK_CUDA_ERROR(hipMalloc(&d_argmin, sizeof(int32_t) + sizeof(DATA_TYPE)));
    // Allocate temporary storage
    void *d_temp_storage = NULL; size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_distances, d_argmin, k);
    CHECK_CUDA_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));
    
    // Run argmin-reduction
    hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_distances + i * k, d_argmin, k);

    int32_t argmin_idx;
    DATA_TYPE argmin_val;
    CHECK_CUDA_ERROR(hipMemcpy(&argmin_idx, &(d_argmin->key), sizeof(int32_t), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&argmin_val, &(d_argmin->value), sizeof(DATA_TYPE), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipFree(d_temp_storage));
    CHECK_CUDA_ERROR(hipFree(d_argmin));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    ++h_clusters_len[argmin_idx];
    h_points_clusters[i] = argmin_idx;
  }
  CHECK_CUDA_ERROR(hipMemcpy(d_points_clusters, h_points_clusters, n * sizeof(uint32_t), hipMemcpyHostToDevice));
}

void schedule_argmin_kernel(const hipDeviceProp_t *props, const uint32_t n, const uint32_t k, dim3 *grid, dim3 *block, uint32_t *warps_per_block, uint32_t *sh_mem) {
  dim3 argmin_grid_dim(n);
  dim3 argmin_block_dim(max(next_pow_2(k), props->warpSize));
  
  *grid   = argmin_grid_dim;
  *block  = argmin_block_dim;
  *warps_per_block = (k + props->warpSize - 1) / props->warpSize; // Ceil
  *sh_mem = (*warps_per_block) * sizeof(Pair);
}