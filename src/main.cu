#include <stdio.h>
#include <vector>
#include <algorithm>
#include <string>
#include <fstream>

#include "../include/common.h"
#include "../include/cxxopts.hpp"
#include "../include/input_parser.hpp"
#include "../include/errors.hpp"
#include "../lib/cuda/utils.cuh"
#include "./kmeans.h"

#define ARG_DIMENSIONS  0
#define ARG_SAMPLES     1
#define ARG_CLUSTERS    2
#define ARG_MAXITER     3
#define ARG_OUTFILE     4
const char* ARG_STR[5] = {"dimensions", "n-samples", "clusters", "maxiter", "out-file"};

using namespace std;

cxxopts::ParseResult args;
int getArg_u (int arg) {
  try {
    return args[ARG_STR[arg]].as<int>();
  } catch(...) {
    printErrDesc(EXIT_ARGS);
    cerr << ARG_STR[arg] << endl;
    exit(EXIT_ARGS);
  }
}
string getArg_s (int arg) {
  try {
    return args[ARG_STR[arg]].as<string>();
  } catch(...) {
    printErrDesc(EXIT_ARGS);
    cerr << ARG_STR[arg] << endl;
    exit(EXIT_ARGS);
  }
}

int main(int argc, char **argv) {
  // Read input args
  cxxopts::Options options("gpukmeans", "gpukmeans is an implementation of the K-means algorithm that uses a GPU");
  
  options.add_options()
    ("h,help", "Print usage")
    ("d,dimensions",  "Number of dimensions of a point",  cxxopts::value<int>())
    ("n,n-samples",   "Number of points",                 cxxopts::value<int>())
    ("k,clusters",    "Number of clusters",               cxxopts::value<int>())
    ("m,maxiter",     "Maximum number of iterations",     cxxopts::value<int>())
    ("o,out-file",    "Output filename",                  cxxopts::value<string>());

  args = options.parse(argc, argv);

  if (args.count("help")) {
    cout << options.help() << endl;
    exit(0);
  }

  unsigned int  d         = getArg_u(ARG_DIMENSIONS);
  size_t        n         = getArg_u(ARG_SAMPLES);
  unsigned int  k         = getArg_u(ARG_CLUSTERS);
  size_t        maxiter   = getArg_u(ARG_MAXITER);
  string        out_file  = getArg_s(ARG_OUTFILE);
  
  InputParser<DATA_TYPE> input(cin, d, n);
  if (DEBUG_INPUT_DATA) cout << input << endl;

  // Check devices
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    printf("hipGetDeviceCount returned %d\n-> %s\n", static_cast<int>(error_id), hipGetErrorString(error_id));
    exit(EXIT_FAILURE);
  }
  if (deviceCount == 0) {
    printErrDesc(EXIT_CUDA_DEV);
    exit(EXIT_CUDA_DEV);
  } else if (DEBUG_DEVICE) {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }

  int dev = 0;
  hipSetDevice(dev); // Use device 0 by default
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  if (DEBUG_DEVICE) describeDevice(dev, deviceProp);
  
  Kmeans kmeans(n, d, k, input.get_dataset());
  uint64_t converged = kmeans.run(maxiter);

  #if DEBUG_OUTPUT_INFO
    if (converged < maxiter)
      printf("K-means converged at iteration %lu\n", converged);
    else
      printf("K-means did NOT converge\n");
  #endif

  ofstream fout(out_file);
  kmeans.to_csv(fout);
  fout.close();

  return 0;
}