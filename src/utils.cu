#include <iostream>

#include "utils.cuh"

using namespace std;

void check(hipError_t err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    cerr << "CUDA Runtime Error at: " << file << ":" << line << endl;
    cerr << hipGetErrorString(err) << " " << func << endl;
    exit(EXIT_FAILURE);
  }
}

void checkLast(const char* const file, const int line) {
  hipError_t err{hipGetLastError()};
  if (err != hipSuccess) {
    cerr << "CUDA Runtime Error at: " << file << ":" << line << endl;
    cerr << hipGetErrorName(err) << ": " << hipGetErrorString(err) << endl;
    exit(EXIT_FAILURE);
  }
}

void describeDevice (int dev, hipDeviceProp_t& deviceProp) {
  printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

  int driverVersion = 0, runtimeVersion = 0;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
          driverVersion / 1000, (driverVersion % 100) / 10,
          runtimeVersion / 1000, (runtimeVersion % 100) / 10);
  printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
          deviceProp.major, deviceProp.minor);
  
  char msg[256];
  snprintf(msg, sizeof(msg),
             "  Total amount of global memory:                 %.0f MBytes "
             "(%llu bytes)\n",
             static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
             (unsigned long long)deviceProp.totalGlobalMem);

  printf("%s", msg);

  printf("  %d multiprocessors\n", deviceProp.multiProcessorCount);
  printf(
      "  GPU Max Clock rate:                            %.0f MHz (%0.2f "
      "GHz)\n",
      deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
  printf("  Memory Clock rate:                             %.0f Mhz\n",
           deviceProp.memoryClockRate * 1e-3f);
  printf("  Memory Bus Width:                              %d-bit\n",
          deviceProp.memoryBusWidth);

  if (deviceProp.l2CacheSize) {
    printf("  L2 Cache Size:                                 %d bytes\n",
            deviceProp.l2CacheSize);
  }

  printf("  Total amount of constant memory:               %zu bytes\n",
           deviceProp.totalConstMem);
  printf("  Total amount of shared memory per block:       %zu bytes\n",
          deviceProp.sharedMemPerBlock);
  printf("  Total shared memory per multiprocessor:        %zu bytes\n",
          deviceProp.sharedMemPerMultiprocessor);
  printf("  Total number of registers available per block: %d\n",
          deviceProp.regsPerBlock);
  printf("  Warp size:                                     %d\n",
          deviceProp.warpSize);
  printf("  Maximum number of threads per multiprocessor:  %d\n",
          deviceProp.maxThreadsPerMultiProcessor);
  printf("  Maximum number of threads per block:           %d\n",
          deviceProp.maxThreadsPerBlock);
  printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
          deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
          deviceProp.maxThreadsDim[2]);
  printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
          deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
          deviceProp.maxGridSize[2]);
  printf("  Maximum memory pitch:                          %zu bytes\n",
          deviceProp.memPitch);
}

unsigned int next_pow_2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}
