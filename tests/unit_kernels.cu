#include "hip/hip_runtime.h"
#include <catch2/catch_test_macros.hpp>

#include <limits>

#include "../src/kernels/kernels.cuh"
#include "../src/utils.cuh"
#include "../src/include/common.h"

#define KERNEL_CENTROIDS 0

const DATA_TYPE infty   = numeric_limits<DATA_TYPE>::infinity();
const DATA_TYPE EPSILON = numeric_limits<DATA_TYPE>::epsilon();

#define TEST_DEBUG 1

TEST_CASE("kernel_distances", "[kernel][distances]") {
  const unsigned int TESTS_N = 8;
  const unsigned int N[TESTS_N] = {10, 10, 17, 51, 159, 1000, 3456, 10056};
  const unsigned int D[TESTS_N] = { 1,  2,  3,  5,  11,   12,   24,    32};
  const unsigned int K[TESTS_N] = { 2,  6,  3, 28,   7,  500, 1763,  9056};

  for (int i = 2; i < 3; ++i) { // FIXME
    const unsigned int n = N[i];
    const unsigned int d = D[i];
    const unsigned int k = K[i];

    char test_name[50];
    sprintf(test_name, "kernel compute_distances_shfl n: %u  d: %u  k: %u", n, d, k);
    SECTION(test_name) {

      DATA_TYPE *h_points = new DATA_TYPE[n * d];
      DATA_TYPE *h_centroids = new DATA_TYPE[k * d];
      DATA_TYPE *h_distances = new DATA_TYPE[n * k];
      for (uint32_t i = 0; i < n; ++i) {
        for (uint32_t j = 0; j < d; ++j) {
          h_points[i * d + j] = std::rand() / 10002.32;
          if (TEST_DEBUG) printf("%.3f, ", h_points[i * d + j]);
        }
        if (TEST_DEBUG) printf("\n");
      }
      if (TEST_DEBUG) printf("\n");
      for (uint32_t i = 0; i < k; ++i) {
        for (uint32_t j = 0; j < d; ++j) {
          h_centroids[i * d + j] = static_cast <DATA_TYPE> (std::rand() / 10002.45);
          if (TEST_DEBUG) printf("%.3f, ", h_points[i * d + j]);
        }
        if (TEST_DEBUG) printf("\n");
      }
      DATA_TYPE *d_distances;
      hipMalloc(&d_distances, sizeof(DATA_TYPE) * n * k);
      DATA_TYPE *d_points;
      hipMalloc(&d_points, sizeof(DATA_TYPE) * n * d);
      hipMemcpy(d_points, h_points, sizeof(DATA_TYPE) * n * d, hipMemcpyHostToDevice);
      DATA_TYPE *d_centroids;
      hipMalloc(&d_centroids, sizeof(DATA_TYPE) * k * d);
      hipMemcpy(d_centroids, h_centroids, sizeof(DATA_TYPE) * k * d, hipMemcpyHostToDevice);

      const uint32_t dist_max_points_per_warp = 32 / next_pow_2(d);
      dim3 dist_grid_dim(ceil(((float) n) / dist_max_points_per_warp), k);
      dim3 dist_block_dim(dist_max_points_per_warp * next_pow_2(d));

      compute_distances_shfl<<<dist_grid_dim, dist_block_dim>>>(d_distances, d_centroids, d_points, n, dist_max_points_per_warp, d, next_pow_2(d));
      hipMemcpy(h_distances, d_distances, sizeof(DATA_TYPE) * n * k,  hipMemcpyDeviceToHost);

      DATA_TYPE* cpu_distances = new DATA_TYPE[n * k];
      for (uint32_t ni = 0; ni < n; ++ni) {
        for (uint32_t ki = 0; ki < k; ++ki) {
          DATA_TYPE dist = 0, tmp;
          for (uint32_t di = 0; di < d; ++di) {
            tmp = h_points[ni * d + di] - h_centroids[ki * d + di];
            dist += tmp * tmp;
          }
          cpu_distances[ni * k + ki] = dist;
        }
      }

      hipDeviceSynchronize();

      for (uint32_t i = 0; i < n * k; ++i) {
        if (TEST_DEBUG) printf("point: %u center: %u cmp: %.6f -- %.6f\n", i / k, i % k, h_distances[i], cpu_distances[i]);
        REQUIRE( h_distances[i] - cpu_distances[i] < EPSILON );
      }

      hipFree(d_distances);
      hipFree(d_points);
      hipFree(d_centroids);
    }
  }
}

TEST_CASE("kernel_argmin", "[kernel][argmin]") {
  const unsigned int TESTS_N = 8;
  const unsigned int N[TESTS_N] = {2, 10, 17, 51, 159, 1000, 3456, 10056};
  const unsigned int K[TESTS_N] = {1,  2,  7,  5, 129,  997, 1023, 1024};

  for (int i = 0; i < TESTS_N; ++i) {
    const unsigned int n = N[i];
    const unsigned int k = K[i];
    const unsigned int SIZE = n * k;

    char test_name[50];
    sprintf(test_name, "kernel clusters_argmin_shfl n: %u  k: %u", n, k);
    SECTION(test_name) {
        
      DATA_TYPE *h_distances = new DATA_TYPE[SIZE];
      for (uint32_t i = 0; i < n; ++i) {
        for (uint32_t j = 0; j < k; ++j) {
          h_distances[i * k + j] = static_cast <DATA_TYPE> (std::rand() / 105.456);
          // printf("%-2u %-2u -> %.0f\n", i, j, h_distances[i * k + j]);
        }
      }
      DATA_TYPE *d_distances;
      hipMalloc(&d_distances, sizeof(DATA_TYPE) * SIZE);
      hipMemcpy(d_distances, h_distances, sizeof(DATA_TYPE) * SIZE,  hipMemcpyHostToDevice);

      uint32_t* d_clusters_len;
      hipMalloc(&d_clusters_len, k * sizeof(uint32_t));
      hipMemset(d_clusters_len, 0, k * sizeof(uint32_t));

      uint32_t* d_points_clusters;
      hipMalloc(&d_points_clusters, sizeof(uint32_t) * n);
      
      uint32_t warps_per_block = (k + 32 - 1) / 32; // Ceil
      clusters_argmin_shfl<<<n, max(next_pow_2(k), 32)>>>(n, k, d_distances, d_points_clusters, d_clusters_len, warps_per_block, infty);
      hipDeviceSynchronize();

      uint32_t h_points_clusters[n];
      hipMemcpy(h_points_clusters, d_points_clusters, sizeof(uint32_t) * n,  hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
      for (uint32_t i = 0; i < n; i++) {
        DATA_TYPE min = infty;
        uint32_t idx = 0;
        for (uint32_t j = 0, ii = i * k; j < k; j++, ii++) {
          // printf("j: %u, ii: %u, v: %.0f\n", j, ii, h_distances[ii]);
          if (h_distances[ii] < min) {
            min = h_distances[ii];
            idx = j;
          }
        }
        
        REQUIRE( h_points_clusters[i] == idx );
        //printf("%-7u -> %5u (should be %-5u %.3f)\n", i, h_points_clusters[i], idx, min);
      }
      hipFree(d_distances);
      hipFree(d_clusters_len);
      hipFree(d_points_clusters);
    }
  }
}

TEST_CASE("kernel centroids", "[kernel][centroids]") {
  #define TESTS_N 8
  const unsigned int D[TESTS_N] = {2,  3,  10,  32,  50,  100, 1000, 1024};
  const unsigned int N[TESTS_N] = {2, 10, 100,  51, 159, 1000, 3456, 10056};
  const unsigned int K[TESTS_N] = {1,  4,   7,  10, 129,  997, 1023, 1024};
  
  for (int d_idx = 0; d_idx < 4; ++d_idx) {
    for (int n_idx = 0; n_idx < TESTS_N; ++n_idx) {
      for (int k_idx = 0; k_idx < TESTS_N; ++k_idx) {
        const unsigned int d = D[d_idx];
        const unsigned int n = N[n_idx];
        const unsigned int k = K[k_idx];
        char test_name[50];
      
        snprintf(test_name, 49, "kernel centroids d=%u n=%u k=%u", d, n, k);

        SECTION(test_name) {
          DATA_TYPE *h_centroids = new DATA_TYPE[k * d];
          DATA_TYPE *h_points = new DATA_TYPE[n * d];
          uint32_t  *h_points_clusters = new uint32_t[n];
          uint32_t  *h_clusters_len = new uint32_t[k];
          
          memset(h_clusters_len, 0, k * sizeof(uint32_t));
          for (uint32_t i = 0; i < n; ++i) {
            h_points_clusters[i] = (static_cast <uint32_t> (std::rand() % k));
            h_clusters_len[h_points_clusters[i]]++;
            for (uint32_t j = 0; j < d; ++j) {
              h_points[i * d + j] = (static_cast <DATA_TYPE> (std::rand() / 1000.0)) / 1000.00;
            }
          }
          
          memset(h_centroids, 0, k * d * sizeof(DATA_TYPE));
          for (uint32_t i = 0; i < n; ++i) {
            for (uint32_t j = 0; j < d; ++j) {
              h_centroids[h_points_clusters[i] * d + j] += h_points[i * d + j];
            }
          }

          for (uint32_t i = 0; i < k; ++i) {
            for (uint32_t j = 0; j < d; ++j) {
              uint64_t count = h_clusters_len[i] > 1 ? h_clusters_len[i] : 1; 
              DATA_TYPE scale = 1.0 / ((double) count); 
              h_centroids[i * d + j] *= scale; 
            }
          }  
          
          dim3 cent_grid_dim(k);
          dim3 cent_block_dim((((int) n) > 32) ? next_pow_2((n + 1) / 2) : 32, d); 
          int cent_threads_tot = cent_block_dim.x * cent_block_dim.y;
          while (cent_threads_tot > 1024) {
            cent_block_dim.x /= 2;
            cent_grid_dim.y *= 2;
            cent_threads_tot = cent_block_dim.x * cent_block_dim.y;
          }

          DATA_TYPE* d_centroids;
          CHECK_CUDA_ERROR(hipMalloc(&d_centroids, k * d * sizeof(DATA_TYPE)));
          DATA_TYPE* d_points;
          CHECK_CUDA_ERROR(hipMalloc(&d_points, n * d * sizeof(DATA_TYPE)));
          CHECK_CUDA_ERROR(hipMemcpy(d_points, h_points, n * d * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
          uint32_t* d_points_clusters;
          CHECK_CUDA_ERROR(hipMalloc(&d_points_clusters, n * sizeof(uint32_t)));
          CHECK_CUDA_ERROR(hipMemcpy(d_points_clusters, h_points_clusters, n * sizeof(uint32_t), hipMemcpyHostToDevice));
          uint32_t* d_clusters_len;
          CHECK_CUDA_ERROR(hipMalloc(&d_clusters_len, k * sizeof(uint32_t)));
          CHECK_CUDA_ERROR(hipMemcpy(d_clusters_len, h_clusters_len, k * sizeof(uint32_t), hipMemcpyHostToDevice));
          
          #if KERNEL_CENTROIDS == 0
            compute_centroids_shfl<<<cent_grid_dim, cent_block_dim>>>(d_centroids, d_points, d_points_clusters, d_clusters_len, n, d);
          #else
            size_t cent_sh_mem = (cent_block_dim.x / 32) * k * d * sizeof(DATA_TYPE);
            compute_centroids_shfl_shrd<<<cent_grid_dim, cent_block_dim, cent_sh_mem>>>(d_centroids, d_points, d_points_clusters, d_clusters_len, n, d);
          #endif
          hipDeviceSynchronize();

          DATA_TYPE *h_centroids_cpy = new DATA_TYPE[k * d];
          CHECK_CUDA_ERROR(hipMemcpy(h_centroids_cpy, d_centroids, k * d * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));

          const DATA_TYPE EPSILON = numeric_limits<DATA_TYPE>::round_error();
          bool is_equal = true;
          for (uint32_t i = 0; i < k; ++i) {
            for (uint32_t j = 0; j < d; ++j) {
              is_equal &= fabs(h_centroids[i * d + j] - h_centroids_cpy[i * d + j]) < EPSILON;
            }
          }  
          
          free(h_centroids);
          free(h_centroids_cpy);
          free(h_points);
          free(h_points_clusters);
          free(h_clusters_len);
          hipFree(d_centroids);
          hipFree(d_points);
          hipFree(d_points_clusters);
          hipFree(d_clusters_len);

          REQUIRE(is_equal);
        }
      }
    }
  }
}
