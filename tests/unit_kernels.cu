#include "hip/hip_runtime.h"
#include <catch2/catch_test_macros.hpp>

#include <limits>

#include "../src/kernels/kernels.cuh"
#include "../src/utils.cuh"
#include "../src/include/common.h"

#define KERNEL_CENTROIDS 0

#define TESTS_N 8
const unsigned int D[TESTS_N] = {2,  3,  10,  32,  50,  100, 1000, 1024};
const unsigned int N[TESTS_N] = {2, 10, 100,  51, 159, 1000, 3456, 10056};
const unsigned int K[TESTS_N] = {1,  4,   7,  10, 129,  997, 1023, 1024};
DATA_TYPE infty = numeric_limits<DATA_TYPE>::infinity();

TEST_CASE("kernel argmin", "[kernel][argmin]") {
  for (int i = 0; i < TESTS_N; ++i) {
    const unsigned int n = N[i];
    const unsigned int k = K[i];
    const unsigned int SIZE = n * k;
    char test_name[20];
    
    snprintf(test_name, 19, "kernel argmin %u %u", n, k);

    SECTION(test_name) {
        
        DATA_TYPE *h_distances = new DATA_TYPE[SIZE];
        for (uint32_t i = 0; i < n; ++i) {
            for (uint32_t j = 0; j < k; ++j) {
                h_distances[i * k + j] = static_cast <DATA_TYPE> (std::rand() / 100.0);
                // printf("%-2u %-2u -> %.0f\n", i, j, h_distances[i * k + j]);
            }
        }
        DATA_TYPE *d_distances;
        hipMalloc((void **)&d_distances, sizeof(DATA_TYPE) * SIZE);
        hipMemcpy(d_distances, h_distances, sizeof(DATA_TYPE) * SIZE,  hipMemcpyHostToDevice);

        uint32_t* d_clusters_len;
        hipMalloc(&d_clusters_len, k * sizeof(uint32_t));
        hipMemset(d_clusters_len, 0, k * sizeof(uint32_t));

        uint32_t* d_points_clusters;
        hipMalloc((void **)&d_points_clusters, sizeof(uint32_t) * n);
       
        uint32_t warps_per_block = (k + 32 - 1) / 32; // Ceil
        clusters_argmin_shfl<<<n, max(next_pow_2(k), 32)>>>(n, k, d_distances, d_points_clusters, d_clusters_len, warps_per_block, infty);
        hipDeviceSynchronize();

        uint32_t h_points_clusters[n];
        hipMemcpy(h_points_clusters, d_points_clusters, sizeof(uint32_t) * n,  hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        for (uint32_t i = 0; i < n; i++) {
          DATA_TYPE min = infty;
          uint32_t idx = 0;
          for (uint32_t j = 0, ii = i * k; j < k; j++, ii++) {
            // printf("j: %u, ii: %u, v: %.0f\n", j, ii, h_distances[ii]);
            if (h_distances[ii] < min) {
              min = h_distances[ii];
              idx = j;
            }
          }
          REQUIRE( h_points_clusters[i] == idx );
          //printf("%-7u -> %5u (should be %-5u %.3f)\n", i, h_points_clusters[i], idx, min);
        }
        
        hipFree(d_clusters_len);
        hipFree(d_distances);
        hipFree(d_points_clusters);
        free(h_distances);
    }
  }
}

TEST_CASE("kernel centroids", "[kernel][centroids]") {
  for (int d_idx = 0; d_idx < 4; ++d_idx) {
    for (int n_idx = 0; n_idx < TESTS_N; ++n_idx) {
      for (int k_idx = 0; k_idx < TESTS_N; ++k_idx) {
        const unsigned int d = D[d_idx];
        const unsigned int n = N[n_idx];
        const unsigned int k = K[k_idx];
        char test_name[50];
      
        snprintf(test_name, 49, "kernel centroids d=%u n=%u k=%u", d, n, k);

        SECTION(test_name) {
          DATA_TYPE *h_centroids = new DATA_TYPE[k * d];
          DATA_TYPE *h_points = new DATA_TYPE[n * d];
          uint32_t  *h_points_clusters = new uint32_t[n];
          uint32_t  *h_clusters_len = new uint32_t[k];
          
          memset(h_clusters_len, 0, k * sizeof(uint32_t));
          for (uint32_t i = 0; i < n; ++i) {
            h_points_clusters[i] = (static_cast <uint32_t> (std::rand() % k));
            h_clusters_len[h_points_clusters[i]]++;
            for (uint32_t j = 0; j < d; ++j) {
              h_points[i * d + j] = (static_cast <DATA_TYPE> (std::rand() / 1000.0)) / 1000.00;
            }
          }
          
          memset(h_centroids, 0, k * d * sizeof(DATA_TYPE));
          for (uint32_t i = 0; i < n; ++i) {
            for (uint32_t j = 0; j < d; ++j) {
              h_centroids[h_points_clusters[i] * d + j] += h_points[i * d + j];
            }
          }

          for (uint32_t i = 0; i < k; ++i) {
            for (uint32_t j = 0; j < d; ++j) {
              uint64_t count = h_clusters_len[i] > 1 ? h_clusters_len[i] : 1; 
              DATA_TYPE scale = 1.0 / ((double) count); 
              h_centroids[i * d + j] *= scale; 
            }
          }  
          
          dim3 cent_grid_dim(k);
          dim3 cent_block_dim((((int) n) > 32) ? next_pow_2((n + 1) / 2) : 32, d); 
          int cent_threads_tot = cent_block_dim.x * cent_block_dim.y;
          while (cent_threads_tot > 1024) {
            cent_block_dim.x /= 2;
            cent_grid_dim.y *= 2;
            cent_threads_tot = cent_block_dim.x * cent_block_dim.y;
          }

          DATA_TYPE* d_centroids;
          CHECK_CUDA_ERROR(hipMalloc(&d_centroids, k * d * sizeof(DATA_TYPE)));
          DATA_TYPE* d_points;
          CHECK_CUDA_ERROR(hipMalloc(&d_points, n * d * sizeof(DATA_TYPE)));
          CHECK_CUDA_ERROR(hipMemcpy(d_points, h_points, n * d * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
          uint32_t* d_points_clusters;
          CHECK_CUDA_ERROR(hipMalloc(&d_points_clusters, n * sizeof(uint32_t)));
          CHECK_CUDA_ERROR(hipMemcpy(d_points_clusters, h_points_clusters, n * sizeof(uint32_t), hipMemcpyHostToDevice));
          uint32_t* d_clusters_len;
          CHECK_CUDA_ERROR(hipMalloc(&d_clusters_len, k * sizeof(uint32_t)));
          CHECK_CUDA_ERROR(hipMemcpy(d_clusters_len, h_clusters_len, k * sizeof(uint32_t), hipMemcpyHostToDevice));
          
          #if KERNEL_CENTROIDS == 0
            compute_centroids_shfl<<<cent_grid_dim, cent_block_dim>>>(d_centroids, d_points, d_points_clusters, d_clusters_len, n, d);
          #else
            size_t cent_sh_mem = (cent_block_dim.x / 32) * k * d * sizeof(DATA_TYPE);
            compute_centroids_shfl_shrd<<<cent_grid_dim, cent_block_dim, cent_sh_mem>>>(d_centroids, d_points, d_points_clusters, d_clusters_len, n, d);
          #endif
          hipDeviceSynchronize();

          DATA_TYPE *h_centroids_cpy = new DATA_TYPE[k * d];
          CHECK_CUDA_ERROR(hipMemcpy(h_centroids_cpy, d_centroids, k * d * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));

          const DATA_TYPE EPSILON = numeric_limits<DATA_TYPE>::round_error();
          bool is_equal = true;
          for (uint32_t i = 0; i < k; ++i) {
            for (uint32_t j = 0; j < d; ++j) {
              is_equal &= fabs(h_centroids[i * d + j] - h_centroids_cpy[i * d + j]) < EPSILON;
            }
          }  
          
          free(h_centroids);
          free(h_centroids_cpy);
          free(h_points);
          free(h_points_clusters);
          free(h_clusters_len);
          hipFree(d_centroids);
          hipFree(d_points);
          hipFree(d_points_clusters);
          hipFree(d_clusters_len);

          REQUIRE(is_equal);
        }
      }
    }
  }
}